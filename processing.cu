#include "hip/hip_runtime.h"
#include "processing.h"
#include "Runnable.h"

namespace processing 
{

	ImageFactory::ImageFactory(const string & fileName):
		d_rGBAPointer_(nullptr),
		d_grayPointer_(nullptr)
	{
		h_imageRGBA_ = cv::imread(fileName, CV_LOAD_IMAGE_COLOR);
		if (h_imageRGBA_.empty()) {
			std::cerr << "Couldn't open file: " << fileName << std::endl;
			exit(1);
		}

		cv::cvtColor(h_imageRGBA_, h_imageRGBA_, CV_BGR2RGBA);
		cv::cvtColor(h_imageRGBA_, h_imageGray_, CV_RGBA2GRAY);

		const size_t numPixels = h_imageRGBA_.rows * h_imageRGBA_.cols;

		d_rGBAPointer_ = allocateMemmoryDevice<uchar4>(numPixels);
		d_grayPointer_ = allocateMemmoryDevice<uchar>(numPixels);

		copyHostRGBAToDeviceRGBA();
		copyHostGrayToDeviceGray();
	}

	ImageFactory::~ImageFactory()
	{
		deallocateMemmoryDevice(d_rGBAPointer_);
		deallocateMemmoryDevice(d_grayPointer_);
	}

	void ImageFactory::copyHostRGBAToDeviceRGBA()
	{
		checkCudaErrors(hipMemcpy(d_rGBAPointer_, getHostRGBAPointer(), h_imageRGBA_.rows * h_imageRGBA_.cols * sizeof(uchar4), hipMemcpyHostToDevice));
	}

	void ImageFactory::copyDeviceRGBAToHostRGBA()
	{
		checkCudaErrors(hipMemcpy(getHostRGBAPointer(), d_rGBAPointer_, h_imageRGBA_.rows * h_imageRGBA_.cols * sizeof(uchar4), hipMemcpyDeviceToHost));
	}

	void ImageFactory::copyHostGrayToDeviceGray()
	{
		checkCudaErrors(hipMemcpy(d_grayPointer_, getHostGrayPointer(), h_imageGray_.rows * h_imageGray_.cols * sizeof(uchar), hipMemcpyHostToDevice));
	}

	void ImageFactory::copyDeviceGrayToHostGray()
	{
		checkCudaErrors(hipMemcpy(getHostGrayPointer(), d_grayPointer_, h_imageGray_.rows * h_imageGray_.cols * sizeof(uchar), hipMemcpyDeviceToHost));
	}

	void ImageFactory::saveRGBAImg(const string & filename)
	{
		imwrite(filename, h_imageRGBA_);
	}

	void ImageFactory::saveGrayImg(const string & filename)
	{
		imwrite(filename, h_imageGray_);
	}

	void ImageFactory::run(Runnable * r)
	{
		r->run(*this);
	}

	void ImageFactory::deallocateMemmoryDevice(void * pointer)
	{
		checkCudaErrors(hipFree(pointer));
	}

}




__global__ void processing::kernels::nullGray(uchar * grayPtr, const size_t numPixels)
{
	for (size_t i = 0; i < numPixels; i++)
	{
		grayPtr[i] = 0;
	}
}
