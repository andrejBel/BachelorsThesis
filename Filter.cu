#include "Filter.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <processing.h>
#include <algorithm>

namespace processing 
{
	template<typename T, int WIDTH>
	Filter<T, WIDTH>::Filter(vector<T> filter, const T multiplier)
	{
		std::copy(filter.data(), filter.data() + WIDTH*WIDTH, filter_);
		if (multiplier != 1.0) 
		{
			for (uint i = 0; i < WIDTH*WIDTH; ++i)
			{
				filter_[i] *= multiplier;
			}
		}
	}

	template<typename T, int WIDTH>
	Filter<T, WIDTH>::Filter(T * filter, const T multiplier)
	{
		std::copy(filter, filter + WIDTH*WIDTH, filter_);
		if (multiplier != 1.0)
		{
			for (uint i = 0; i < WIDTH*WIDTH; ++i)
			{
				filter_[i] *= multiplier;
			}
		}
	}

	template<typename T, int WIDTH>
	CPU void Filter<T, WIDTH>::copyWholeFilterToDeviceMemory(void * destination) const
	{
		checkCudaErrors(hipMemcpy(destination, this, sizeof(Filter<T, WIDTH>), hipMemcpyHostToDevice));
	}



}
