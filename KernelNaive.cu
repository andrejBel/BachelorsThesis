#include "hip/hip_runtime.h"
#include "KernelNaive.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "opencv2/core/utility.hpp"

#include <vector>
#include <cstdio>
#include <cmath>
#include <iostream>

#include <thread>
#include <algorithm>


using namespace std;




namespace processing
{

#define CONVOLUTIONSLOWNAIVE(FILTERWIDTH)\
case FILTERWIDTH:\
{\
	float * ptr =  (deviceFilters.get() + offset);\
	convolutionGPUNaive << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGrayImageIn.get(), deviceGrayImageOut.get(), FILTERWIDTH);\
	break;\
}

	
	__global__ void convolutionGPUNaive(float * filter, const int numRows, const int numCols, float * inputImage, float * outputImage, int filterWidth)
	{
		int2 absoluteImagePosition;

		absoluteImagePosition.x = blockIdx.x * blockDim.x + threadIdx.x;
		absoluteImagePosition.y = blockIdx.y * blockDim.y + threadIdx.y;
		if (absoluteImagePosition.x >= numCols || absoluteImagePosition.y >= numRows)
		{
			return;
		}
		const size_t index1D = absoluteImagePosition.y * numCols + absoluteImagePosition.x;
		float result(0.0);
		int2 pointPosition;
		for (int yOffset = 0; yOffset < filterWidth; yOffset++)
		{
			for (int xOffset = 0; xOffset < filterWidth; xOffset++)
			{
				pointPosition.x = absoluteImagePosition.x + xOffset - filterWidth / 2;
				pointPosition.y = absoluteImagePosition.y + yOffset - filterWidth / 2;
				pointPosition.x = KernelNaive::min(KernelNaive::max(pointPosition.x, 0), numCols - 1);
				pointPosition.y = KernelNaive::min(KernelNaive::max(pointPosition.y, 0), numRows - 1);
				result += filter[yOffset*filterWidth + xOffset] * inputImage[pointPosition.y*numCols + pointPosition.x];
			}
		}
		outputImage[index1D] = result;
	}


	KernelNaive::KernelNaive()
	{
	}


	void KernelNaive::run(ImageFactory& image, vector<shared_ptr<AbstractFilter>>& filters, vector<shared_ptr<float>>& results)
	{
		shared_ptr<float> deviceFilters = makeDeviceFilters(filters);
		
		// filter allocation and initialization
		shared_ptr<float> deviceGrayImageOut = allocateMemmoryDevice<float>(image.getNumPixels());
		const float * hostGrayImage = image.getInputGrayPointerFloat();

		shared_ptr<float> deviceGrayImageIn = allocateMemmoryDevice<float>(image.getNumPixels());
		checkCudaErrors(hipMemcpy(deviceGrayImageIn.get(), hostGrayImage, image.getNumPixels() * sizeof(float), hipMemcpyHostToDevice));
		// memory allocation

		const uint numberOfThreadsInBlock = 16;
		const dim3 blockSize(numberOfThreadsInBlock, numberOfThreadsInBlock);
		const dim3 gridSize((image.getNumCols() + blockSize.x - 1) / blockSize.x, (image.getNumRows() + blockSize.y - 1) / blockSize.y, 1);
		// kernels parameters
		uint offset(0);
		for (auto& filter : filters)
		{
			switch (filter->getWidth())
			{
			case 1:
			{
				float * ptr = (deviceFilters.get() + offset);

				convolutionGPUNaive << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGrayImageIn.get(), deviceGrayImageOut.get(), 1); 
				break; 
			}
				//CONVOLUTIONSLOWNAIVE(1)
				CONVOLUTIONSLOWNAIVE(3)
				CONVOLUTIONSLOWNAIVE(5)
				CONVOLUTIONSLOWNAIVE(7)
				CONVOLUTIONSLOWNAIVE(9)
				CONVOLUTIONSLOWNAIVE(11)
				CONVOLUTIONSLOWNAIVE(13)
				CONVOLUTIONSLOWNAIVE(15)
			default:
				std::cerr << "Filter with width: " << filter->getWidth() << " not supported!" << endl;
				break;
			}
			offset += filter->getSize();
			shared_ptr<float> resultCPU = makeArray<float>(image.getNumPixels());
			checkCudaErrors(hipMemcpy(resultCPU.get(), deviceGrayImageOut.get(), image.getNumPixels() * sizeof(float), hipMemcpyDeviceToHost));
			results.push_back(resultCPU);
		}
		checkCudaErrors(hipDeviceSynchronize());
	}

}
