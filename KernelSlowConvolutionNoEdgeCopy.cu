#include "hip/hip_runtime.h"
#include "KernelSlowConvolutionNoEdgeCopy.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "opencv2/core/utility.hpp"

#include <vector>
#include <cstdio>
#include <cmath>
#include <iostream>

#include <thread>
#include <algorithm>

using namespace std;




namespace processing
{

#define CONVOLUTIONSLOWNOEDGECOPY(FILTERWIDTH)\
case FILTERWIDTH:\
{\
	float * ptr =  (deviceFilters.get() + offset);\
	convolutionGPU<FILTERWIDTH> << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGrayImageIn.get(), deviceGrayImageOut.get());\
	break;\
}

	template<typename int FILTER_WIDTH>
	__global__ void convolutionGPU(float * filter, const int numRows, const int numCols, float * inputImage, float * outputImage)
	{
		int2 absoluteImagePosition;

		absoluteImagePosition.x = blockIdx.x * blockDim.x + threadIdx.x;
		absoluteImagePosition.y = blockIdx.y * blockDim.y + threadIdx.y;
		if (absoluteImagePosition.x >= numCols || absoluteImagePosition.y >= numRows)
		{
			return;
		}
		const size_t index1D = absoluteImagePosition.y * numCols + absoluteImagePosition.x;
		float result(0.0);
		int2 pointPosition;
		#pragma unroll FILTER_WIDTH
		for (int yOffset = 0; yOffset < FILTER_WIDTH; yOffset++)
		{
		#pragma unroll FILTER_WIDTH
			for (int xOffset = 0; xOffset < FILTER_WIDTH; xOffset++)
			{
				pointPosition.x = absoluteImagePosition.x + xOffset - FILTER_WIDTH / 2;
				pointPosition.y = absoluteImagePosition.y + yOffset - FILTER_WIDTH / 2;
				pointPosition.x = min(max(pointPosition.x, 0), numCols - 1);
				pointPosition.y = min(max(pointPosition.y, 0), numRows - 1);
				result += filter[yOffset*FILTER_WIDTH + xOffset] * inputImage[pointPosition.y*numCols + pointPosition.x];
			}
		}
		outputImage[index1D] = result;

	}


	__host__ __forceinline__ int KernelSlowConvolutionNoEdgeCopy::indexToCopyToMirrored(int index, int numCols, int numRows, const int filterWidth)
	{
		int indexX = (index % (numCols + (filterWidth / 2) * 2)) - (filterWidth / 2);
		int indexY = (index / (numCols + (filterWidth / 2) * 2)) - (filterWidth / 2);
		indexX = std::min(std::max(indexX, 0), numCols - 1);
		indexY = std::min(std::max(indexY, 0), numRows - 1);
		return indexY * numCols + indexX;
	}



	KernelSlowConvolutionNoEdgeCopy::KernelSlowConvolutionNoEdgeCopy() 
	{
	}


	void KernelSlowConvolutionNoEdgeCopy::run(ImageFactory& image, vector<shared_ptr<Filter>>& filters, vector<shared_ptr<float>>& results)
	{
		shared_ptr<float> deviceFilters = makeDeviceFilters(filters);
		// filter allocation and initialization
		shared_ptr<float> deviceGrayImageOut = allocateMemmoryDevice<float>(image.getNumPixels());
		const float* hostGrayImage = image.getInputGrayPointerFloat();

		shared_ptr<float> deviceGrayImageIn = allocateMemmoryDevice<float>(image.getNumPixels());
		checkCudaErrors(hipMemcpy(deviceGrayImageIn.get(), hostGrayImage, image.getNumPixels() * sizeof(float), hipMemcpyHostToDevice));
		// memory allocation

		const uint numberOfThreadsInBlock = 16;
		const dim3 blockSize(numberOfThreadsInBlock, numberOfThreadsInBlock);
		const dim3 gridSize((image.getNumCols() + blockSize.x - 1) / blockSize.x, (image.getNumRows() + blockSize.y - 1) / blockSize.y, 1);
		// kernels parameters
		uint offset = 0;
		for (auto& filter : filters)
		{
			switch (filter->getWidth())
			{
			CONVOLUTIONSLOWNOEDGECOPY(1)
			CONVOLUTIONSLOWNOEDGECOPY(3)
			CONVOLUTIONSLOWNOEDGECOPY(5)
			CONVOLUTIONSLOWNOEDGECOPY(7)
			CONVOLUTIONSLOWNOEDGECOPY(9)
			CONVOLUTIONSLOWNOEDGECOPY(11)
			CONVOLUTIONSLOWNOEDGECOPY(13)
			CONVOLUTIONSLOWNOEDGECOPY(15)
			default:
				std::cerr << "Filter with width: " << filter->getWidth() << " not supported!" << endl;
				break;
			}
			offset += filter->getSize();
			shared_ptr<float> resultCPU = makeArray<float>(image.getNumPixels());
			checkCudaErrors(hipDeviceSynchronize());
			checkCudaErrors(hipMemcpy(resultCPU.get(), deviceGrayImageOut.get(), image.getNumPixels() * sizeof(float), hipMemcpyDeviceToHost));
			results.push_back(resultCPU);		
		}
	}
	
}
