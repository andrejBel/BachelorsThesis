#include "hip/hip_runtime.h"
#include "KernelSlowConvolutionNoEdgeCopy.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "processing.h"
#include "Filter.h"

#include "opencv2/core/utility.hpp"

#include <vector>
#include <cstdio>
#include <cmath>
#include <iostream>

#include <thread>
#include <algorithm>

using namespace std;

template <typename T>
__global__ void separateChannels(const uchar4* const inputImageRGBA, int numRows, int numCols, unsigned char* const redChannel, unsigned char* const greenChannel, unsigned char* const blueChannel)
{
	int absolute_image_position_x = blockIdx.x * blockDim.x + threadIdx.x;
	int absolute_image_position_y = blockIdx.y * blockDim.y + threadIdx.y;
	if (absolute_image_position_x >= numCols || absolute_image_position_y >= numRows)
	{
		return;
	}
	const int index_1D = absolute_image_position_y * numCols + absolute_image_position_x;
	redChannel[index_1D] = inputImageRGBA[index_1D].x;
	greenChannel[index_1D] = inputImageRGBA[index_1D].y;
	blueChannel[index_1D] = inputImageRGBA[index_1D].z;
}

template <typename T>
__global__ void recombineChannels(const unsigned char* const redChannel, const unsigned char* const greenChannel, const unsigned char* const blueChannel, uchar4* const outputImageRGBA, int numRows, int numCols)
{
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;
	outputImageRGBA[thread_1D_pos] = make_uchar4(redChannel[thread_1D_pos], greenChannel[thread_1D_pos], blueChannel[thread_1D_pos], 255);
}

namespace processing
{

	template <typename T>
	__device__ __forceinline__ const T min(const T a, const T b) {
		return !(b<a) ? a : b;
	}

	template <typename T>
	__device__  __forceinline__ const T max(const T a, const T b) {
		return (b<a) ? a : b;
	}

	__device__ __forceinline__ size_t indexInNew(int indexX, int indexY, int originalWidth, int originalHeight, int filterWidth)
	{
		int newWidth = originalWidth + (filterWidth / 2) * 2;
		indexX += filterWidth / 2;
		indexY += filterWidth / 2;
		return indexY * newWidth + indexX;
	}



	template<typename T, typename int FILTER_WIDTH>
	__global__ void convolutionGPU(processing::Filter<T, FILTER_WIDTH> * filter, const int numRows, const int numCols, uchar * inputImage, T * outputImage)
	{
		int2 absoluteImagePosition;

		absoluteImagePosition.x = blockIdx.x * blockDim.x + threadIdx.x;
		absoluteImagePosition.y = blockIdx.y * blockDim.y + threadIdx.y;
		if (absoluteImagePosition.x >= numCols || absoluteImagePosition.y >= numRows)
		{
			return;
		}
		const size_t index1D = absoluteImagePosition.y * numCols + absoluteImagePosition.x;
		const T* filterV = filter->getFilter();
		T result(0.0);
		int2 pointPosition;
		#pragma unroll FILTER_WIDTH
		for (int yOffset = 0; yOffset < FILTER_WIDTH; yOffset++)
		{
		#pragma unroll FILTER_WIDTH
			for (int xOffset = 0; xOffset < FILTER_WIDTH; xOffset++)
			{
				pointPosition.x = absoluteImagePosition.x + xOffset - FILTER_WIDTH / 2;
				pointPosition.y = absoluteImagePosition.y + yOffset - FILTER_WIDTH / 2;
				pointPosition.x = min(max(pointPosition.x, 0), numCols - 1);
				pointPosition.y = min(max(pointPosition.y, 0), numRows - 1);
				result += filterV[yOffset*FILTER_WIDTH + xOffset] * inputImage[pointPosition.y*numCols + pointPosition.x];
			}
		}
		outputImage[index1D] = result;

	}


	template<typename T>
	__host__ __forceinline__ int KernelSlowConvolutionNoEdgeCopy<T>::indexToCopyToMirrored(int index, int numCols, int numRows, const int filterWidth)
	{
		int indexX = (index % (numCols + (filterWidth / 2) * 2)) - (filterWidth / 2);
		int indexY = (index / (numCols + (filterWidth / 2) * 2)) - (filterWidth / 2);
		indexX = std::min(std::max(indexX, 0), numCols - 1);
		indexY = std::min(std::max(indexY, 0), numRows - 1);
		return indexY * numCols + indexX;
	}



	template<typename T>
	KernelSlowConvolutionNoEdgeCopy<T>::KernelSlowConvolutionNoEdgeCopy(vector<shared_ptr<AbstractFilter<T>>>& filters) :
		h_filters_(filters),
		threadPool_(1)
	{
	}

	template<typename T>
	void KernelSlowConvolutionNoEdgeCopy<T>::run(ImageFactory & image, vector<shared_ptr<T>>& results)
	{
		uint filterCount(h_filters_.size());
		size_t memmoryToAllocateForFiltersOnDevice(0);
		for_each(h_filters_.begin(), h_filters_.end(), [&memmoryToAllocateForFiltersOnDevice](auto& filter) { memmoryToAllocateForFiltersOnDevice += filter->getSize(); });
		shared_ptr<uchar> deviceFilters = allocateMemmoryDevice<uchar>(memmoryToAllocateForFiltersOnDevice);
		uint offset(0);
		int maxFilterWidth = 0;
		for_each(h_filters_.begin(), h_filters_.end(), [&deviceFilters, &offset, &maxFilterWidth](auto& filter)
		{
			filter->copyWholeFilterToDeviceMemory(deviceFilters.get() + offset);
			offset += filter->getSize();
			if (maxFilterWidth < filter->getSize())
			{
				maxFilterWidth = filter->getSize();
			}
		});
		// filter allocation and initialization
		shared_ptr<T> deviceGrayImageOut = allocateMemmoryDevice<T>(image.getNumPixels());
		uchar* hostGrayImage = image.getInputGrayPointer();

		shared_ptr<uchar> deviceGrayImageIn = allocateMemmoryDevice<uchar>(image.getNumPixels());
		shared_ptr<T> result = makeArrayCudaHost<T>(image.getNumPixels());
		checkCudaErrors(hipMemcpy(deviceGrayImageIn.get(), hostGrayImage, image.getNumPixels() * sizeof(uchar), hipMemcpyHostToDevice));
		// memory allocation

		const uint numberOfThreadsInBlock = 16;
		const dim3 blockSize(numberOfThreadsInBlock, numberOfThreadsInBlock);
		const dim3 gridSize((image.getNumCols() + blockSize.x - 1) / blockSize.x, (image.getNumRows() + blockSize.y - 1) / blockSize.y, 1);
		// kernels parameters
		offset = 0;
		for (auto& filter : h_filters_)
		{
			switch (filter->getWidth())
			{
			case 3:
			{
				Filter<T, 3> * ptr = (Filter<T, 3> *) (deviceFilters.get() + offset);
				convolutionGPU << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGrayImageIn.get(), deviceGrayImageOut.get());
				checkCudaErrors(hipDeviceSynchronize());
				break;
			}
			case 5:
			{
				Filter<T, 5> * ptr = (Filter<T, 5> *) (deviceFilters.get() + offset);
				convolutionGPU << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGrayImageIn.get(), deviceGrayImageOut.get());
				checkCudaErrors(hipDeviceSynchronize());

				break;
			}
			case 7:
			{
				Filter<T, 7> * ptr = (Filter<T, 7> *) (deviceFilters.get() + offset);
				convolutionGPU << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGrayImageIn.get(), deviceGrayImageOut.get());
				checkCudaErrors(hipDeviceSynchronize());
				break;
			}
			default:
				break;
			}
			offset += filter->getSize();
			threadPool_.finishAll();
			checkCudaErrors(hipMemcpy(result.get(), deviceGrayImageOut.get(), image.getNumPixels() * sizeof(T), hipMemcpyDeviceToHost));
			threadPool_.addTask(
				[&] ()
				{
				shared_ptr<T> resultCPU = makeArray<T>(image.getNumPixels());
				std::copy(result.get(), result.get() + image.getNumPixels(), resultCPU.get());
				results.push_back(resultCPU);
				}
			);
			
			//image.copyDeviceGrayToHostGrayOut(deviceGrayImageOut.get());
			//image.saveGrayImgOut("blurredImage.jpg");
		}
		cout << "";
		threadPool_.finishAll();
	}
	
}
