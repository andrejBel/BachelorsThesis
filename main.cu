#include "hip/hip_runtime.h"
#pragma once
#include <stdio.h>
#include <iostream>

#include "processing.h"

#include <vector>
#include <memory>

//#include "KernelSlowConvolution.cu"
#include "KernelSlowConvolutionNoEdgeCopy.cu"
#include "KernelSharedMemory.cu"
#include "Filter.cu"
#include "CpuSlowConvolution.cpp"
#include <vector>
#include <algorithm>

using namespace cv;
using namespace std;
using namespace processing;

static const string INPUT_IMAGE_PATH = "input_img.jpg";
static const string OUTPUT_IMAGE_PATH = "output_img.jpg";



int main()
{
	allocateMemmoryDevice<uchar>(1);
	#define COUNT_TYPE float
	auto gausianBig = createFilter<COUNT_TYPE>(
		7, {
			0.00000067f,0.00002292f,0.00019117f,0.00038771f,	0.00019117f,	0.00002292f,	0.00000067f,
			0.00002292f,	0.00078634f,	0.00655965f,	0.01330373f,	0.00655965f,	0.00078633f,	0.00002292f,
			0.00019117f,	0.00655965f,	0.05472157f,	0.11098164f,	0.05472157f,	0.00655965f,	0.00019117f,
			0.00038771f,	0.01330373f,	0.11098164f,	0.22508352f,	0.11098164f,	0.01330373f,	0.00038771f,
			0.00019117f,	0.00655965f,	0.05472157f,	0.11098164f,	0.05472157f,	0.00655965f,	0.00019117f,
			0.00002292f,	0.00078633f,	0.00655965f,	0.01330373f,	0.00655965f,	0.00078633f, 0.00002292f,
			0.00000067f,	0.00002292f,	0.00019117f,	0.00038771f,	0.00019117f,	0.00002292f,	0.00000067f
		},
		1.0f
		);
	auto gausianBlur = createFilter<COUNT_TYPE>
		(3,
		{
			0.0f,0.2f,0.0f,
			0.2f,0.2f,0.2f,
			0.0f, 0.2f, 0.0f
		}
			
	);
	auto sobel = createFilter<COUNT_TYPE>
	(3, 
	{
		-10.0f,8.4f,11.0f,
		 2.0f,4.3f,22.0f,
		-1.0f,2.5f,12.0f
	}, 10.0f
	);
	auto someFilter = createFilter<COUNT_TYPE>
		(5,
		{
			-1.0f,2.0f,1.0f,1.5f, 1.6f,
			-2.0f,7.0f,2.0f,5.8f,12.1f,
			-1.0f,8.0f,2.0f,4.7f,3.3f
		}, 1.7f
			);
	vector< shared_ptr<AbstractFilter<COUNT_TYPE> > > filters = { gausianBig };
	for (size_t i = 0; i < gausianBlur->getWidth() * gausianBlur->getWidth(); i++)
	{
		cout << ((Filter<COUNT_TYPE, 3> *) gausianBlur.get())->getFilter()[i] << endl;
	}
	KernelSlowConvolutionNoEdgeCopy<COUNT_TYPE> kernelNormal(filters);
	KernelSharedMemory<COUNT_TYPE> kernel(filters);
	CpuSlowConvolution<COUNT_TYPE> cpuKonv(filters);
	ImageFactory factory(INPUT_IMAGE_PATH);
	
	vector<shared_ptr<COUNT_TYPE>> resultsKernelNormal;
	auto timeGPU1 = factory.run(&kernelNormal, resultsKernelNormal);
	vector<shared_ptr<COUNT_TYPE>> resultsKernelShare;
	auto timeGPU = factory.run(&kernel, resultsKernelShare);
	vector<shared_ptr<COUNT_TYPE>> resultsCpu;
	auto timeCPU = factory.run(&cpuKonv, resultsCpu);
	cout << "konec" ;
	cout << "Time GPU normal: " << timeGPU1.getTimeMilli() << endl;
	cout << "Time GPU shared: " << timeGPU.getTimeMilli() << endl;
	cout << "Time CPU: " << timeCPU.getTimeMilli() << endl;
	for (uint i = 0; i < std::min(resultsKernelShare.size(), resultsCpu.size()) ; i++)
	{
		auto pGPU = resultsKernelShare[i];
		auto pCPU = resultsCpu[i];
		for (size_t i = 0; i < factory.getNumPixels(); i++) 
		{
			if ( [&pGPU, &pCPU, i] () {
				return fabs(pGPU.get()[i] - pCPU.get()[i]) > 0.1;
			}() )
			{
				cout << "-----------------------" << endl;
				cout << "Index: " << i << endl;
				cout << "GPU " << pGPU.get()[i] << endl;
				cout << "CPU " << pCPU.get()[i] << endl;
				cout << "-----------------------" << endl;
			}
		}
	}
	//factory.saveRGBAImg(OUTPUT_IMAGE_PATH);

	//imwrite(OUTPUT_IMAGE_PATH, factory.getHostGray());
	return 0;
}