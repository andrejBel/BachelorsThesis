#include "hip/hip_runtime.h"
#pragma once
#include <stdio.h>
#include <iostream>

#include "processing.h"

#include <vector>
#include <memory>
#include "KernelSlowConvolution.cu"
#include "Filter.cu"
#include "CpuSlowConvolution.cpp"
#include <vector>
#include <algorithm>

using namespace cv;
using namespace std;
using namespace processing;

static const string INPUT_IMAGE_PATH = "input_img.jpg";
static const string OUTPUT_IMAGE_PATH = "output_img.jpg";



int main()
{
	allocateMemmoryDevice<uchar>(1);
	#define COUNT_TYPE double
	auto gausianBig = createFilter<COUNT_TYPE>(
		7, {
			0.00000067,	0.00002292,	0.00019117,	0.00038771,	0.00019117,	0.00002292,	0.00000067,
			0.00002292,	0.00078634,	0.00655965,	0.01330373,	0.00655965,	0.00078633,	0.00002292,
			0.00019117,	0.00655965,	0.05472157,	0.11098164,	0.05472157,	0.00655965,	0.00019117,
			0.00038771,	0.01330373,	0.11098164,	0.22508352,	0.11098164,	0.01330373,	0.00038771,
			0.00019117,	0.00655965,	0.05472157,	0.11098164,	0.05472157,	0.00655965,	0.00019117,
			0.00002292,	0.00078633,	0.00655965,	0.01330373,	0.00655965,	0.00078633, 0.00002292,
			0.00000067,	0.00002292,	0.00019117,	0.00038771,	0.00019117,	0.00002292,	0.00000067
		},
		1.0
		);
	auto gausianBlur = createFilter<COUNT_TYPE>
		(3,
		{
			0,0.2,0,
			0.2,0.2,0.2,
			0, 0.2, 0
		}
			
	);
	auto sobel = createFilter<COUNT_TYPE>
	(3, 
	{
		-1.0,0.0,1,
		-2.0,0.0,2.0,
		-1.0,0.0,1.0
	}, 10.0
	);
	vector< shared_ptr<AbstractFilter<COUNT_TYPE> > > filters = {  sobel, gausianBlur, gausianBig };
	for (size_t i = 0; i < gausianBlur->getWidth() * gausianBlur->getWidth(); i++)
	{
		cout << ((Filter<COUNT_TYPE, 3> *) gausianBlur.get())->getFilter()[i] << endl;
	}
	KernelSlowConvolution<COUNT_TYPE> kernel(filters);
	CpuSlowConvolution<COUNT_TYPE> cpuKonv(filters);
	ImageFactory factory(INPUT_IMAGE_PATH);
	
	vector<shared_ptr<COUNT_TYPE>> resultsKernel;
	auto timeGPU = factory.run(&kernel, resultsKernel);
	vector<shared_ptr<COUNT_TYPE>> resultsCpu;
	auto timeCPU = factory.run(&cpuKonv, resultsCpu);
	cout << "konec" ;
	cout << "Time GPU: " << timeGPU.getTimeMicro() << endl;
	cout << "Time CPU: " << timeCPU.getTimeMicro() << endl;
	for (uint i = 0; i < std::min(resultsKernel.size(), resultsCpu.size()) ; i++)
	{
		auto pGPU = resultsKernel[i];
		auto pCPU = resultsCpu[i];
		for (size_t i = 0; i < factory.getNumPixels(); i++) 
		{
			if ( [&pGPU, &pCPU, i] () {
				return fabs(pGPU.get()[i] - pCPU.get()[i]) > 0.001;
			}() )
			{
				cout << "-----------------------" << endl;
				cout << "Index: " << i << endl;
				cout << "GPU " << pGPU.get()[i] << endl;
				cout << "CPU " << pCPU.get()[i] << endl;
				cout << "-----------------------" << endl;
			}
		}
	}
	//factory.saveRGBAImg(OUTPUT_IMAGE_PATH);

	//imwrite(OUTPUT_IMAGE_PATH, factory.getHostGray());
	return 0;
}