#include "hip/hip_runtime.h"
#include "KernelSharedThreadsMulti.h"
#include "KernelSharedThreads.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "opencv2/core/utility.hpp"


#include <vector>
#include <cstdio>
#include <cmath>
#include <iostream>

#include <algorithm>
#include <type_traits>
#include <utility>
#include "MemoryPoolPinned.h"
#include "MemoryPoolPitched.h"
#include <algorithm>
#include <thread>
#include <mutex>
#include <condition_variable>
#include <stack>
#include <queue>
#include <map>
#include <utility>
#include "ThreadPool.h"

#define MUL(a, b) __mul24(a, b)
#define IMAD(a, b, c) ( __mul24((a), (b)) + (c) )
#define CEIL(a, b) ((a + b - 1) / b)
#define ROUNDUP(a, b) (((a + b - 1) / b) * b)

using namespace std;
namespace processing
{

#define CONVOLUTIONSHAREDMULTIFULLBLOCK(FILTERWIDTH, BLOCKSIZEX, BLOCKSIZEY) \
						case FILTERWIDTH: \
						{ \
							hipMemcpyToSymbolAsync(HIP_SYMBOL(FILTERCUDA), job.filters_.get(), sizeof(float) * FILTERWIDTH * FILTERWIDTH * job.filterCount_, 0, hipMemcpyHostToDevice, stream.stream_); \
							const short MAX_SMALL_TILE_DIMENION_X = 3; \
							const short MAX_SMALL_TILE_DIMENION_Y = 3; \
							int colsForGridX = CEIL(job.numCols, MAX_SMALL_TILE_DIMENION_X); \
							int rowsForGridY = CEIL(job.numRows, MAX_SMALL_TILE_DIMENION_Y); \
							const int FILTER_WIDTH = FILTERWIDTH; \
							const int BLOCK_SIZE_X = BLOCKSIZEX; \
							const int BLOCK_SIZE_Y = BLOCKSIZEY; \
							const dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y); \
							const dim3 gridSize((colsForGridX + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (rowsForGridY + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, 1); \
							if (job.makeZeros) \
							{ \
								switch (job.filterCount_) \
								{ \
								case 1: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 1, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, true, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 2: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 2, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 3: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 3, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 4: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 4, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 5: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 5, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 6: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 6, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 7: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 7, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 8: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 8, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 9: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 9, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 10: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 10, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								} \
							} \
							else  \
							{ \
								switch (job.filterCount_) \
								{ \
								case 1: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 1, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, false, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 2: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 2, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 3: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 3, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 4: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 4, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break;  \
								case 5: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 5, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 6: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 6, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 7: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 7, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 8: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 8, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 9: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 9, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 10: convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, 10, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								} \
							} \
							break; \
						}

#define CONVOLUTIONSHAREDTHREADSSMALLMULTI(FILTERWIDTH, BLOCKSIZEX, BLOCKSIZEY, TILESIZEX, TILESIZEY) \
						case FILTERWIDTH: \
						{ \
							hipMemcpyToSymbolAsync(HIP_SYMBOL(FILTERCUDA), job.filters_.get(), sizeof(float) * FILTERWIDTH * FILTERWIDTH * job.filterCount_, 0, hipMemcpyHostToDevice, stream.stream_); \
							const short MAX_SMALL_TILE_DIMENION_X = 2; \
							const short MAX_SMALL_TILE_DIMENION_Y = 2; \
							int colsForGridX = CEIL(job.numCols, MAX_SMALL_TILE_DIMENION_X); \
							int rowsForGridY = CEIL(job.numRows, MAX_SMALL_TILE_DIMENION_Y); \
							const int FILTER_WIDTH = FILTERWIDTH; \
							const int BLOCK_SIZE_X = BLOCKSIZEX; \
							const int BLOCK_SIZE_Y = BLOCKSIZEY; \
							const int TILE_SIZE_X = TILESIZEX; \
							const int TILE_SIZE_Y = TILESIZEY; \
							const dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y); \
							const dim3 gridSize((colsForGridX + TILE_SIZE_X - 1) / TILE_SIZE_X, (rowsForGridY + TILE_SIZE_Y - 1) / TILE_SIZE_Y, 1); \
							if (job.makeZeros) \
							{ \
								switch (job.filterCount_) \
								{ \
								case 1: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 1, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, true, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 2: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 2, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 3: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 3, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 4: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 4, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 5: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 5, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 6: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 6, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 7: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 7, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 8: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 8, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 9: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 9, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 10: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 10, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, true> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								} \
							} \
							else  \
							{ \
								switch (job.filterCount_) \
								{ \
								case 1: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 1, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, false, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 2: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 2, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 3: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 3, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 4: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 4, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break;  \
								case 5: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 5, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 6: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 6, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 7: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 7, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 8: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 8, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 9: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 9, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								case 10: convolutionGPUSharedIncompleteBlockSmall< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, 10, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, false> << <gridSize, blockSize, 0, stream.stream_ >> > (job.inputImage_, pitchInput_ / sizeof(float), pitchOutput_ / sizeof(float), job.bufferStart_); break; \
								} \
							} \
							break; \
						}


	namespace static_if_detail {

		struct identity {
			template<typename T>
			__device__ T operator()(T&& x) const {
				return std::forward<T>(x);
			}
		};

		template<bool Cond>
		struct statement {
			template<typename F>
			__device__ void then(const F& f) {
				f(identity());
			}

			template<typename F>
			__device__ void else_(const F&) {}
		};

		template<>
		struct statement<false> {
			template<typename F>
			__device__ void then(const F&) {}

			template<typename F>
			__device__ void else_(const F& f) {
				f(identity());
			}
		};

	} //end of namespace static_if_detail



	template<bool Cond, typename F>
	__device__ static_if_detail::statement<Cond> static_if(F const& f) {
		static_if_detail::statement<Cond> if_;
		if_.then(f);
		return if_;
	}

	template<typename int FILTER_WIDTH, typename int BLOCK_SIZE_X, typename int BLOCK_SIZE_Y, typename int TILE_SIZE_X, typename int TILE_SIZE_Y, typename int FILTER_COUNT, typename int BUFFER_SIZE, typename bool MAKEZEROS, typename bool SYNC = true, typename int MAX_SMALL_TILE_DIMENION_X = 2, typename int MAX_SMALL_TILE_DIMENION_Y = 2>
	__global__ void convolutionGPUSharedIncompleteBlockSmall(const float * __restrict__ inputImage, const int inputPitch, const int outputPitch, const short bufferStartPosition)
	{
		__shared__ float shared[BLOCK_SIZE_Y * MAX_SMALL_TILE_DIMENION_Y][BLOCK_SIZE_X * MAX_SMALL_TILE_DIMENION_X];
		float results[MAX_SMALL_TILE_DIMENION_X * MAX_SMALL_TILE_DIMENION_Y];
		int threadX = MUL(threadIdx.x, MAX_SMALL_TILE_DIMENION_X);
		int threadY = MUL(threadIdx.y, MAX_SMALL_TILE_DIMENION_Y);
		int2 absoluteImagePosition;
		absoluteImagePosition.x = IMAD(blockIdx.x, TILE_SIZE_X, threadIdx.x) * MAX_SMALL_TILE_DIMENION_X;
		absoluteImagePosition.y = IMAD(blockIdx.y, TILE_SIZE_Y, threadIdx.y) * MAX_SMALL_TILE_DIMENION_Y;

#pragma unroll MAX_SMALL_TILE_DIMENION_Y
		for (int i = 0; i < MAX_SMALL_TILE_DIMENION_Y; i++)
		{
			*((float2 *)&shared[threadY + i][threadX]) = *(float2 *)(inputImage + IMAD(absoluteImagePosition.y + i, inputPitch, absoluteImagePosition.x));
		}
		__syncthreads();
		if (threadX < TILE_SIZE_X * MAX_SMALL_TILE_DIMENION_X  && threadY < TILE_SIZE_Y * MAX_SMALL_TILE_DIMENION_Y)
		{
			float * outputImage;
			float filterValue;
#pragma unroll FILTER_COUNT
			for (int i = 0; i < FILTER_COUNT; ++i)
			{
				outputImage = PITCHED_MEMORY_BUFFER_DEVICE.memory_[(bufferStartPosition + i) % BUFFER_SIZE];
#pragma unroll MAX_SMALL_TILE_DIMENION_Y
				for (int k = 0; k < MAX_SMALL_TILE_DIMENION_Y; k++)
				{
#pragma unroll MAX_SMALL_TILE_DIMENION_X
					for (int l = 0; l < MAX_SMALL_TILE_DIMENION_X; l++)
					{
						results[k * MAX_SMALL_TILE_DIMENION_Y + l] = 0.0;
					}
				}
#pragma unroll FILTER_WIDTH
				for (int yOffset = 0; yOffset < FILTER_WIDTH; yOffset++)
				{
#pragma unroll FILTER_WIDTH
					for (int xOffset = 0; xOffset < FILTER_WIDTH; xOffset++)
					{
						filterValue = FILTERCUDA[yOffset*FILTER_WIDTH + xOffset + FILTER_WIDTH * FILTER_WIDTH * i];
#pragma unroll MAX_SMALL_TILE_DIMENION_Y
						for (int i = 0; i < MAX_SMALL_TILE_DIMENION_Y; i++)
						{
#pragma unroll MAX_SMALL_TILE_DIMENION_X
							for (int j = 0; j < MAX_SMALL_TILE_DIMENION_X; j++)
							{
								results[IMAD(i, MAX_SMALL_TILE_DIMENION_Y, j)] += filterValue * shared[yOffset + threadY + i][xOffset + threadX + j];
							}
						}
					}
				}
				/*
				#pragma unroll MAX_SMALL_TILE_DIMENION_Y
				for (int i = 0; i < MAX_SMALL_TILE_DIMENION_Y; i++)
				{
				#pragma unroll MAX_SMALL_TILE_DIMENION_X
				for (int j = 0; j < MAX_SMALL_TILE_DIMENION_X; j++)
				{
				static_if<MAKEZEROS>([outputImage, absoluteImagePosition, outputPitch, &results, i, j, MAX_SMALL_TILE_DIMENION_X = MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y = MAX_SMALL_TILE_DIMENION_Y](auto f) {
				outputImage[IMAD(absoluteImagePosition.y + i, outputPitch, absoluteImagePosition.x + j)] = results[i *MAX_SMALL_TILE_DIMENION_Y + j];
				}).else_([outputImage, absoluteImagePosition, outputPitch, &results, i, j, MAX_SMALL_TILE_DIMENION_X = MAX_SMALL_TILE_DIMENION_X, MAX_SMALL_TILE_DIMENION_Y = MAX_SMALL_TILE_DIMENION_Y](auto f) {
				outputImage[IMAD(absoluteImagePosition.y + i, outputPitch, absoluteImagePosition.x + j)] += results[i *MAX_SMALL_TILE_DIMENION_Y + j];
				});
				}
				}

				*/
				if (MAKEZEROS)
				{
					*((float2 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x))) = *((float2*)&results[0]);
					*((float2 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x))) = *((float2*)&results[2]);
				}
				else
				{
					float2 fromGlobal = *((float2 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x)));
					fromGlobal.x += results[0];
					fromGlobal.y += results[1];
					*((float2 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x))) = fromGlobal;
					fromGlobal = *((float2 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x)));
					fromGlobal.x += results[2];
					fromGlobal.y += results[3];
					*((float2 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x))) = fromGlobal;
				}
				/*
				static_if<MAKEZEROS>([outputImage, absoluteImagePosition, outputPitch, &results](auto f) {
				*((float2 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x))) = *((float2*)&results[0]);
				*((float2 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x))) = *((float2*)&results[2]);
				}).else_([outputImage, absoluteImagePosition, outputPitch, &results](auto f) {
				float2 fromGlobal = *((float2 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x)));
				fromGlobal.x += results[0];
				fromGlobal.y += results[1];
				*((float2 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x))) = fromGlobal;
				fromGlobal = *((float2 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x)));
				fromGlobal.x += results[2];
				fromGlobal.y += results[3];
				*((float2 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x))) = fromGlobal;
				});
				*/

				static_if<SYNC>([](auto f) {
					__syncthreads();
				});

			}
		}
	}


	template<typename int FILTER_WIDTH, typename int BLOCK_SIZE_X, typename int BLOCK_SIZE_Y, typename int FILTER_COUNT, typename int BUFFER_SIZE, typename int TILE_SIZE_X, typename int TILE_SIZE_Y, typename bool MAKEZEROS, typename bool SYNC = true>
	__global__ void convolutionGPUSharedFullBlock(const float * __restrict__ inputImage, const int inputPitch, const int outputPitch, const short bufferStartPosition)
	{
		float results[TILE_SIZE_X * TILE_SIZE_Y];
		int2 absoluteImagePosition;
		absoluteImagePosition.x = (blockIdx.x *  BLOCK_SIZE_X + threadIdx.x) * TILE_SIZE_X;
		absoluteImagePosition.y = (blockIdx.y * BLOCK_SIZE_Y + threadIdx.y) * TILE_SIZE_Y;
		int2 positionShared;
		positionShared.x = blockIdx.x * BLOCK_SIZE_X * TILE_SIZE_X;
		positionShared.y = blockIdx.y * BLOCK_SIZE_Y * TILE_SIZE_Y;
		__shared__ float shared[BLOCK_SIZE_Y * TILE_SIZE_Y + (FILTER_WIDTH - 1) + TILE_SIZE_Y][BLOCK_SIZE_X *TILE_SIZE_X + (FILTER_WIDTH - 1) + TILE_SIZE_X];
		int threadX = threadIdx.x * TILE_SIZE_X;
		int threadY = threadIdx.y * TILE_SIZE_Y;
		for (int j = threadY; j < FILTER_WIDTH - 1 + BLOCK_SIZE_Y * TILE_SIZE_Y; j += BLOCK_SIZE_Y * TILE_SIZE_Y)
		{
			for (int i = threadX; i < FILTER_WIDTH - 1 + BLOCK_SIZE_X * TILE_SIZE_X; i += BLOCK_SIZE_X * TILE_SIZE_X)
			{
#pragma unroll TILE_SIZE_Y
				for (int k = 0; k < TILE_SIZE_Y; k++)
				{
					
					*((float3 *)&shared[j + k][i]) = *(float3 *)(inputImage + IMAD(positionShared.y + j + k, inputPitch, positionShared.x + i));
				}

			}
		}
		__syncthreads();
		float * outputImage;
		float filterValue;

#pragma unroll FILTER_COUNT
		for (int i = 0; i < FILTER_COUNT; ++i)
		{
			outputImage = PITCHED_MEMORY_BUFFER_DEVICE.memory_[(bufferStartPosition + i) % BUFFER_SIZE];
#pragma unroll TILE_SIZE_Y
			for (int k = 0; k < TILE_SIZE_Y; k++)
			{
#pragma unroll TILE_SIZE_X
				for (int l = 0; l < TILE_SIZE_X; l++)
				{
					results[k * TILE_SIZE_Y + l] = 0.0;
				}
			}
#pragma unroll FILTER_WIDTH
			for (int yOffset = 0; yOffset < FILTER_WIDTH; yOffset++)
			{
#pragma unroll FILTER_WIDTH
				for (int xOffset = 0; xOffset < FILTER_WIDTH; xOffset++)
				{
					filterValue = FILTERCUDA[yOffset*FILTER_WIDTH + xOffset + FILTER_WIDTH * FILTER_WIDTH * i];
#pragma unroll TILE_SIZE_Y
					for (int k = 0; k < TILE_SIZE_Y; k++)
					{
#pragma unroll TILE_SIZE_X
						for (int l = 0; l < TILE_SIZE_X; l++)
						{
							results[k * TILE_SIZE_Y + l] += filterValue * shared[yOffset + threadY + k][xOffset + threadX + l];
						}
					}
				}
			}

			if (MAKEZEROS)
			{
				*((float3 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x))) = *((float3*)&results[0]);
				*((float3 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x))) = *((float3*)&results[3]);
				*((float3 *)(outputImage + IMAD(absoluteImagePosition.y + 2, outputPitch, absoluteImagePosition.x))) = *((float3*)&results[6]);
			}
			else
			{
				float3 fromGlobal = *((float3 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x)));
				fromGlobal.x += results[0];
				fromGlobal.y += results[1];
				fromGlobal.z += results[2];
				*((float3 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x))) = fromGlobal;
				fromGlobal = *((float3 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x)));
				fromGlobal.x += results[3];
				fromGlobal.y += results[4];
				fromGlobal.z += results[5];
				*((float3 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x))) = fromGlobal;
				fromGlobal = *((float3 *)(outputImage + IMAD(absoluteImagePosition.y + 2, outputPitch, absoluteImagePosition.x)));
				fromGlobal.x += results[6];
				fromGlobal.y += results[7];
				fromGlobal.z += results[8];
				*((float3 *)(outputImage + IMAD(absoluteImagePosition.y + 2, outputPitch, absoluteImagePosition.x))) = fromGlobal;
			}
			static_if<SYNC>([](auto f) {
				__syncthreads();
			});
		}
	}

	namespace KernelSharedThreadsMultiNS
	{
		// variables
		CudaStream streams[3];
		std::queue<Job> jobsInProcess_;
		queue<Job> jobsInPostProcess_;
		std::stack<float *> inputImages_;
		bool preprocessPrepared_ = false;

		size_t pitchInput_;
		size_t pitchOutput_;

		mutex mutexInputImages_;
		mutex mutexJobInProcess_;
		mutex mutexProcessPostProcess_;

		condition_variable conditionVariable_;


		void printJob(Job& job)
		{
			cout << "-------------------------------------" << endl;
			cout << "Filter count: " << job.filterCount_ << endl;
			cout << "Buffer start: " << job.bufferStart_ << endl;
			cout << "Filter group start index: " << job.filterGroupStartIndex << endl;
			cout << "Filters: " << job.filters_ << endl;
			cout << "Filter width: " << job.filterWidth_ << endl;
			cout << "Finish: " << job.finish_ << endl;
			cout << "Go to postprocess: " << job.goToPostprocess << endl;
			cout << "Input image: " << job.inputImage_ << endl;
			cout << "Return image: " << job.returnInputImage_ << endl;
			cout << "Make zeros: " << job.makeZeros << endl;
			cout << "Num cols: " << job.numCols << endl;
			cout << "Num rows: " << job.numRows << endl;
			cout << "*************************************" << endl;
		}


		void preprocess(CudaStream& stream, vector<shared_ptr<ImageFactory>>& images, vector<vector<shared_ptr<Filter>>>& filters)
		{
			int maxImageWidth = 0;
			int maxImageHeight = 0;
			for_each(images.begin(), images.end(), [&maxImageWidth, &maxImageHeight](shared_ptr<ImageFactory> image)
			{
				if (image->getNumCols() > maxImageWidth)
				{
					maxImageWidth = image->getNumCols();
				}
				if (image->getNumRows() > maxImageHeight)
				{
					maxImageHeight = image->getNumRows();
				}
			});
			MemoryPoolPitched::getMemoryPoolPitchedForInput().realoc(maxImageWidth, maxImageHeight);
			MemoryPoolPitched::getMemoryPoolPitchedForOutput().realoc(maxImageWidth, maxImageHeight);
			for (int i = 0; i < PITCHED_MEMORY_BUFFER_SIZE_OUTPUT; i++)
			{
				PITCHED_MEMORY_BUFFER_HOST.memory_[i] = MemoryPoolPitched::getMemoryPoolPitchedForOutput().getMemory()[i];
				//cout << "Index host: " << i << ", value: " << PITCHED_MEMORY_BUFFER_HOST.memory_[i] << endl;
			}
			hipMemcpyToSymbolAsync(HIP_SYMBOL(PITCHED_MEMORY_BUFFER_DEVICE.memory_), PITCHED_MEMORY_BUFFER_HOST.memory_, sizeof(float**) * PITCHED_MEMORY_BUFFER_SIZE_OUTPUT, 0, hipMemcpyHostToDevice, stream.stream_);

			vector<float *> inputImagesMemory = MemoryPoolPitched::getMemoryPoolPitchedForInput().getMemory();

			for (float * memory : inputImagesMemory)
			{
				inputImages_.push(memory);
			}
			/*
			for (size_t i = 0; i < inputImagesMemory.size(); i++)
			{
			cout << i << ": " << inputImagesMemory[i] << endl;
			}
			*/
			pitchInput_ = MemoryPoolPitched::getMemoryPoolPitchedForInput().getPitch();
			pitchOutput_ = MemoryPoolPitched::getMemoryPoolPitchedForOutput().getPitch();

			// roztriedenie po sirkach filtra
			const int numCols = images[0]->getNumCols();
			const int numRows = images[0]->getNumRows();

			const size_t imageSize = images.size();
			const size_t groupSize = filters.size();



			int batchSize = 10;
			/*
			if (groupSize <= 8)
			{
				batchSize = 4;
			}
			else if (groupSize <= 10)
			{
				batchSize = 5;
			}
			else
			{
				batchSize = 10;
			}
			*/
			batchSize = std::min(batchSize, PITCHED_MEMORY_BUFFER_SIZE_OUTPUT / 2);


			vector<vector<shared_ptr<Filter>>> filtersCopy(filters);
			std::sort(filtersCopy.begin(), filtersCopy.end(), [](vector<shared_ptr<Filter>>& first, vector<shared_ptr<Filter>> second)
			{
				return first[0]->getWidth() < second[0]->getWidth();
			});
			vector<BatchGroup> filterGroups;
			for (size_t filterGroupIndex = 0; filterGroupIndex < groupSize; filterGroupIndex += batchSize)
			{
				int startOfGroup = filterGroupIndex;
				int endOfGroup = std::min(filterGroupIndex + batchSize - 1, groupSize - 1);
				int totalGroupSize = endOfGroup - startOfGroup + 1;
				BatchGroup group;
				group.filterCount_ = totalGroupSize;
				group.filterStart_ = startOfGroup;
				group.filterEnd_ = endOfGroup;
				map<short, short> filterWidths;
				for (int k = startOfGroup; k <= endOfGroup; k++)
				{
					if (filterWidths.find(filtersCopy[k][0]->getWidth()) == filterWidths.end()) // neobsahuje 
					{
						filterWidths[filtersCopy[k][0]->getWidth()] = 1;
					}
					else // obsahuje
					{
						filterWidths[filtersCopy[k][0]->getWidth()] += 1;
					}
				}
				for (std::pair<int, int> filterSet : filterWidths)
				{
					group.filterInfos_.push_back(BatchGroupInfo(filterSet.first, filterSet.second));
				}
				filterGroups.push_back(std::move(group));
			}
			for (BatchGroup& group : filterGroups)
			{
				int bufferOutputStart = PITCHED_MEMORY_BUFFER_HOST.acquire(group.filterCount_);
				for (int imageIndex = 0; imageIndex < imageSize; ++imageIndex)
				{
					shared_ptr<ImageFactory>& image = images[imageIndex];
					float * hostGrayImage = image->getInputGrayPointerFloat();
					unique_lock<mutex> lock(mutexInputImages_);
					while (inputImages_.empty())
					{
						conditionVariable_.wait(lock);
					}
					float * deviceGrayImageIn = inputImages_.top();
					inputImages_.pop();
					lock.unlock();
					checkCudaErrors(hipMemcpy2DAsync(deviceGrayImageIn, pitchInput_, hostGrayImage, numCols * sizeof(float), numCols * sizeof(float), numRows, hipMemcpyHostToDevice, stream.stream_));
					checkCudaErrors(hipStreamSynchronize(stream.stream_));
					int filterGroupIndexStart = group.filterStart_;
					int localBufferOutputStart = bufferOutputStart;
					vector<Job> jobs;
					for (BatchGroupInfo& info : group.filterInfos_)
					{
						Job job;
						job.numCols = numCols;
						job.numRows = numRows;
						job.inputImage_ = deviceGrayImageIn;
						job.filterCount_ = info.filterCount_;
						job.filterWidth_ = info.filterWidth_;
						job.bufferStart_ = localBufferOutputStart;
						localBufferOutputStart += info.filterCount_;

						int sizeOfFilter = job.filterWidth_* job.filterWidth_;
						shared_ptr<float> memoryForFilters = shared_ptr<float>(new float[sizeOfFilter * job.filterCount_], [](float * ptr) { delete[] ptr; });
						for (int offset = 0, index = filterGroupIndexStart; index < filterGroupIndexStart + job.filterCount_; ++index, offset += sizeOfFilter)
						{
							std::copy(filtersCopy[index][imageIndex]->getFilter(), filtersCopy[index][imageIndex]->getFilter() + sizeOfFilter, memoryForFilters.get() + offset);
						}
						job.filterGroupStartIndex = filterGroupIndexStart;
						filterGroupIndexStart += job.filterCount_;
						job.filters_ = memoryForFilters;


						if (imageIndex == 0) // prvy prvok, nuluje buffer output
						{
							job.makeZeros = true;
						}
						if (&info == &group.filterInfos_[group.filterInfos_.size() - 1]) // posledna grupa, vracia vstupnz obrayok
						{
							job.returnInputImage_ = true;
						}
						if (imageIndex == imageSize - 1) // tento posiela do postprocessingu
						{
							job.goToPostprocess = true;
						}
						jobs.push_back(std::move(job));
					}
					mutexJobInProcess_.lock();
					for (int j = 0; j < jobs.size(); j++)
					{
						if (imageIndex == imageSize - 1 && j == jobs.size() - 1 && &group == &filterGroups[filterGroups.size() - 1])
						{
							jobs[j].finish_ = true;
						}
						//printJob(jobs[j]);
						jobsInProcess_.push(std::move(jobs[j]));
					}

					preprocessPrepared_ = true;
					mutexJobInProcess_.unlock();
					conditionVariable_.notify_all();
				}
			}
			//cout << "Koniec preprocessing" << endl;
		}

		void process(CudaStream& stream)
		{
			queue<Job> jobs;
			bool end = false;
			while (end == false)
			{
				unique_lock<mutex> lock(mutexJobInProcess_);
				while (preprocessPrepared_ == false)
				{
					conditionVariable_.wait(lock);
				}
				if (jobs.size() != 0)
				{
					cout << "Mistake, jobs are not 0!!!!!" << endl;
				}
				std::swap(jobs, jobsInProcess_);
				if (jobs.size() == 0)
				{
					cout << "Mistake jobs connot be 0!!!!!" << endl;
				}
				preprocessPrepared_ = false;
				lock.unlock();
				while (jobs.size())
				{
					Job job = std::move(jobs.front());
					jobs.pop();

					switch (job.filterWidth_)
					{
						CONVOLUTIONSHAREDTHREADSSMALLMULTI(1,32,16,32,16)
						CONVOLUTIONSHAREDTHREADSSMALLMULTI(3, 32, 16, 31, 15)
						CONVOLUTIONSHAREDTHREADSSMALLMULTI(5, 32, 16, 30, 14)
						CONVOLUTIONSHAREDTHREADSSMALLMULTI(7, 32, 32, 29, 29)
						CONVOLUTIONSHAREDMULTIFULLBLOCK(9, 32, 8)
						CONVOLUTIONSHAREDMULTIFULLBLOCK(11, 32, 8)
						CONVOLUTIONSHAREDMULTIFULLBLOCK(13, 32, 6)
						CONVOLUTIONSHAREDMULTIFULLBLOCK(15, 32, 8)
					default:
						std::cerr << "Filter with width: " << job.filterWidth_ << " not supported!" << endl;
						break;
					}

					checkCudaErrors(hipStreamSynchronize(stream.stream_));
					if (job.goToPostprocess)
					{
						mutexProcessPostProcess_.lock();
						jobsInPostProcess_.push(job);
						mutexProcessPostProcess_.unlock();
						conditionVariable_.notify_all();
					}
					if (job.returnInputImage_)
					{
						mutexInputImages_.lock();
						inputImages_.push(job.inputImage_);
						mutexInputImages_.unlock();
						conditionVariable_.notify_all();
					}
					if (job.finish_)
					{
						end = true;
					}
				}
			}
		}

		void postprocess(CudaStream& stream, vector<shared_ptr<float>>& results, size_t filterGroupSize, int numberOfCols, int numberOfRows)
		{
			bool end = false;
			Job job;
			queue<Job> jobs;
			results.resize(filterGroupSize);

			const size_t pixels = numberOfCols * numberOfRows;
			while (end == false)
			{
				unique_lock<mutex> lock(mutexProcessPostProcess_);
				while (jobsInPostProcess_.empty())
				{
					conditionVariable_.wait(lock);
				}
				std::swap(jobs, jobsInPostProcess_);
				lock.unlock();

				while (jobs.size())
				{
					job = std::move(jobs.front());
					jobs.pop();

					int xlen = job.numCols - (job.filterWidth_ - 1);
					int ylen = job.numRows - (job.filterWidth_ - 1);
					const size_t pixels = xlen * ylen;
					for (int i = 0; i < job.filterCount_; i++)
					{

						int filterGroupIndex = job.filterGroupStartIndex + i;
						results[filterGroupIndex] = MemoryPoolPinned::getMemoryPoolPinnedForOutput().acquireMemory(pixels, false);
						checkCudaErrors(hipMemcpy2DAsync(results[filterGroupIndex].get(), xlen * sizeof(float), PITCHED_MEMORY_BUFFER_HOST.memory_[(job.bufferStart_ + i) % PITCHED_MEMORY_BUFFER_SIZE_OUTPUT], pitchOutput_, xlen * sizeof(float), ylen, hipMemcpyDeviceToHost, stream.stream_));
						checkCudaErrors(hipStreamSynchronize(stream.stream_));
						PITCHED_MEMORY_BUFFER_HOST.release(1);
					}
					if (job.finish_)
					{
						end = true;
					}
				}
			}
		}
	}

	KernelSharedThreadsMulti::KernelSharedThreadsMulti() : MultiRunnable()
	{
	}

	void KernelSharedThreadsMulti::run(vector<shared_ptr<ImageFactory>>& images, vector<vector<shared_ptr<Filter>>>& filters, vector<shared_ptr<float>>& results)
	{
		using namespace KernelSharedThreadsMultiNS;

		
		thread threadPreprocessing(preprocess, std::ref(streams[0]), std::ref(images), std::ref(filters));
		thread threadProcessing(process, std::ref(streams[1]));
		thread threadPostprocessing(postprocess, std::ref(streams[2]), std::ref(results), filters.size(), images[0]->getNumCols(), images[0]->getNumRows());
		threadPreprocessing.join();
		threadProcessing.join();
		threadPostprocessing.join();
	}

}