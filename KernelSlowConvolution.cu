#include "hip/hip_runtime.h"
#include "KernelSlowConvolution.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "processing.h"
#include "Filter.h"

#include "opencv2/core/utility.hpp"

#include <vector>
#include <cstdio>
#include <cmath>
#include <iostream>

#include <thread>
using namespace std;

template <typename T>
__global__ void separateChannels(const uchar4* const inputImageRGBA, int numRows, int numCols, unsigned char* const redChannel, unsigned char* const greenChannel, unsigned char* const blueChannel)
{
	int absolute_image_position_x = blockIdx.x * blockDim.x + threadIdx.x;
	int absolute_image_position_y = blockIdx.y * blockDim.y + threadIdx.y;
	if (absolute_image_position_x >= numCols || absolute_image_position_y >= numRows)
	{
		return;
	}
	const int index_1D = absolute_image_position_y * numCols + absolute_image_position_x;
	redChannel[index_1D] = inputImageRGBA[index_1D].x;
	greenChannel[index_1D] = inputImageRGBA[index_1D].y;
	blueChannel[index_1D] = inputImageRGBA[index_1D].z;
}

template <typename T>
__global__ void recombineChannels(const unsigned char* const redChannel, const unsigned char* const greenChannel, const unsigned char* const blueChannel, uchar4* const outputImageRGBA, int numRows, int numCols)
{
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;
	outputImageRGBA[thread_1D_pos] = make_uchar4(redChannel[thread_1D_pos], greenChannel[thread_1D_pos], blueChannel[thread_1D_pos], 255);
}

namespace processing
{

	template <typename T>
	__device__ const T& min(const T& a, const T& b) {
		return !(b<a) ? a : b;
	}

	template <typename T>
	__device__ const T& max(const T& a, const T& b) {
		return (b<a) ? a : b;
	}

	template<typename T, typename int WIDTH>
	__global__ void convolutionGPU(processing::Filter<T, WIDTH> * filter, const int numRows, const int numCols, uchar * inputImage, T * outputImage)
	{
		int2 absoluteImagePosition;

		absoluteImagePosition.x = blockIdx.x * blockDim.x + threadIdx.x;
		absoluteImagePosition.y = blockIdx.y * blockDim.y + threadIdx.y;
		if (absoluteImagePosition.x >= numCols || absoluteImagePosition.y >= numRows)
		{
			return;
		}
		const size_t index1D = absoluteImagePosition.y * numCols + absoluteImagePosition.x;
		const T* filterV = filter->getFilter();
		T result(0.0);
		int2 pointPosition;
		//if (index1D == (1628490))
		//{
#pragma unroll
		for (int yOffset = 0; yOffset < WIDTH; yOffset++)
		{
#pragma unroll
			for (int xOffset = 0; xOffset < WIDTH; xOffset++)
			{

				pointPosition.x = absoluteImagePosition.x + xOffset - WIDTH / 2;
				pointPosition.y = absoluteImagePosition.y + yOffset - WIDTH / 2;
				pointPosition.x = min(max(pointPosition.x, 0), numCols - 1);
				pointPosition.y = min(max(pointPosition.y, 0), numRows - 1);
				result += filterV[yOffset*WIDTH + xOffset] * inputImage[pointPosition.y * numCols + pointPosition.x];
				//printf("Result: %f\n", result);
			}
		}

		outputImage[index1D] = result;
		//}

	}


	template<typename T>
	KernelSlowConvolution<T>::KernelSlowConvolution(vector<shared_ptr<AbstractFilter<T>>>& filters) :
		h_filters_(filters)
	{

	}

	template<typename T>
	void KernelSlowConvolution<T>::run(ImageFactory & image, vector<shared_ptr<T>>& results)
	{
		uint filterCount(h_filters_.size());
		size_t memmoryToAllocateForFiltersOnDevice(0);
		for_each(h_filters_.begin(), h_filters_.end(), [&memmoryToAllocateForFiltersOnDevice](auto& filter) { memmoryToAllocateForFiltersOnDevice += filter->getSize(); });
		shared_ptr<uchar> deviceFilters = allocateMemmoryDevice<uchar>(memmoryToAllocateForFiltersOnDevice);
		uint offset(0);
		for_each(h_filters_.begin(), h_filters_.end(), [&deviceFilters, &offset](auto& filter)
		{
			filter->copyWholeFilterToDeviceMemory(deviceFilters.get() + offset);
			offset += filter->getSize();
		});
		// filter allocation and initialization
		shared_ptr<uchar> deviceGrayImageIn = allocateMemmoryDevice<uchar>(image.getNumPixels());
		shared_ptr<T> deviceGrayImageOut = allocateMemmoryDevice<T>(image.getNumPixels());
		uchar* hostGrayImage = image.getInputGrayPointer();
		checkCudaErrors(hipMemcpy(deviceGrayImageIn.get(), hostGrayImage, image.getNumPixels() * sizeof(uchar), hipMemcpyHostToDevice));
		// memory allocation

		const uint numberOfThreadsInBlock = 32;
		const dim3 blockSize(numberOfThreadsInBlock, numberOfThreadsInBlock);
		const dim3 gridSize((image.getNumCols() + blockSize.x - 1) / blockSize.x, (image.getNumRows() + blockSize.y - 1) / blockSize.y, 1);
		// kernels parameters
		offset = 0;


		for (auto& filter : h_filters_)
		{
			switch (filter->getWidth())
			{
			case 3:
			{
				Filter<T, 3> * ptr = (Filter<T, 3> *) (deviceFilters.get() + offset);
				convolutionGPU << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGrayImageIn.get(), deviceGrayImageOut.get());
				checkCudaErrors(hipDeviceSynchronize());
				break;
			}
			case 5:
			{
				Filter<T, 5> * ptr = (Filter<T, 5> *) (deviceFilters.get() + offset);
				convolutionGPU << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGrayImageIn.get(), deviceGrayImageOut.get());
				checkCudaErrors(hipDeviceSynchronize());

				break;
			}
			case 7:
			{
				Filter<T, 7> * ptr = (Filter<T, 7> *) (deviceFilters.get() + offset);
				convolutionGPU << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGrayImageIn.get(), deviceGrayImageOut.get());
				checkCudaErrors(hipDeviceSynchronize());
				break;
			}
			default:
				break;
			}
			offset += filter->getSize();
			shared_ptr<T> result = makeArray<T>(image.getNumPixels());
			checkCudaErrors(hipMemcpy(result.get(), deviceGrayImageOut.get(), image.getNumPixels() * sizeof(T), hipMemcpyDeviceToHost));
			results.push_back(result);
			//image.copyDeviceGrayToHostGrayOut(deviceGrayImageOut.get());
			//image.saveGrayImgOut("blurredImage.jpg");
		}
	}



	/* color
	template<typename T>
	void KernelSlowConvolution<T>::run(ImageFactory & image, vector<shared_ptr<T>>& results)
	{
	uint filterCount(h_filters_.size());
	size_t memmoryToAllocateForFiltersOnDevice(0);
	for_each(h_filters_.begin(), h_filters_.end(), [&memmoryToAllocateForFiltersOnDevice](auto& filter) { memmoryToAllocateForFiltersOnDevice += filter->getSize(); });
	shared_ptr<uchar> deviceFilters = allocateMemmoryDevice<uchar>( memmoryToAllocateForFiltersOnDevice);
	uint offset(0);
	for_each(h_filters_.begin(), h_filters_.end(), [&deviceFilters, &offset](auto& filter)
	{
	filter->copyWholeFilterToDeviceMemory(deviceFilters.get() + offset);
	offset += filter->getSize();
	});
	// filter allocation and initialization
	shared_ptr<uchar4> deviceRGBAImage = allocateMemmoryDevice<uchar4>(image.getNumPixels());

	shared_ptr<uchar> deviceRedChannelIn = allocateMemmoryDevice<uchar>(image.getNumPixels());
	shared_ptr<uchar> deviceGreenChannelIn = allocateMemmoryDevice<uchar>(image.getNumPixels());
	shared_ptr<uchar> deviceBlueChannelIn = allocateMemmoryDevice<uchar>(image.getNumPixels());

	shared_ptr<uchar> deviceRedChannelOut = allocateMemmoryDevice<uchar>(image.getNumPixels());
	shared_ptr<uchar> deviceGreenChannelOut = allocateMemmoryDevice<uchar>(image.getNumPixels());
	shared_ptr<uchar> deviceBlueChannelOut = allocateMemmoryDevice<uchar>(image.getNumPixels());

	uchar4* hostRGBAImage = image.getInputRGBAPointer();
	checkCudaErrors(hipMemcpy(deviceRGBAImage.get(), hostRGBAImage, image.getNumPixels() * sizeof(uchar4), hipMemcpyHostToDevice));
	// memory allocation
	const uint numberOfThreadsInBlock = 32;
	const dim3 blockSize(numberOfThreadsInBlock, numberOfThreadsInBlock);
	const dim3 gridSize((image.getNumCols() + blockSize.x - 1) / blockSize.x, (image.getNumRows() + blockSize.y - 1) / blockSize.y, 1);
	// kernels parameters
	separateChannels<char> << <gridSize, blockSize >> > (deviceRGBAImage.get(), image.getNumRows(), image.getNumCols(), deviceRedChannelIn.get(), deviceGreenChannelIn.get(), deviceBlueChannelIn.get());
	checkCudaErrors(hipDeviceSynchronize());
	// initialization of channels
	offset = 0;
	for (auto& filter : h_filters_)
	{
	switch (filter->getWidth())
	{
	case 3:
	{
	Filter<T, 3> * ptr = (Filter<T, 3> *) (deviceFilters.get() + offset);
	convolution <<<gridSize, blockSize>>>(ptr, image.getNumRows(), image.getNumCols(), deviceRedChannelIn.get(), deviceRedChannelOut.get());
	checkCudaErrors(hipDeviceSynchronize());
	convolution << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGreenChannelIn.get(), deviceGreenChannelOut.get());
	checkCudaErrors(hipDeviceSynchronize());
	convolution << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceBlueChannelIn.get(), deviceBlueChannelOut.get());
	checkCudaErrors(hipDeviceSynchronize());
	break;
	}
	case 5:
	{
	Filter<T, 5> * ptr = (Filter<T, 5> *) (deviceFilters.get() + offset);
	convolution << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceRedChannelIn.get(), deviceRedChannelOut.get());
	checkCudaErrors(hipDeviceSynchronize());
	convolution << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGreenChannelIn.get(), deviceGreenChannelOut.get());
	checkCudaErrors(hipDeviceSynchronize());
	convolution << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceBlueChannelIn.get(), deviceBlueChannelOut.get());
	checkCudaErrors(hipDeviceSynchronize());
	break;
	}
	case 7:
	{
	Filter<T, 7> * ptr = (Filter<T, 7> *) (deviceFilters.get() + offset);
	convolution << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceRedChannelIn.get(), deviceRedChannelOut.get());
	checkCudaErrors(hipDeviceSynchronize());
	convolution << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGreenChannelIn.get(), deviceGreenChannelOut.get());
	checkCudaErrors(hipDeviceSynchronize());
	convolution << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceBlueChannelIn.get(), deviceBlueChannelOut.get());
	checkCudaErrors(hipDeviceSynchronize());
	break;
	}
	default:
	break;
	}
	offset += filter->getSize();
	recombineChannels<char><<<gridSize, blockSize>>>(deviceRedChannelOut.get(), deviceGreenChannelOut.get(), deviceBlueChannelOut.get(), deviceRGBAImage.get(), image.getNumRows(), image.getNumCols());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors( hipMemcpy(image.getOutputRGBAPointer(), deviceRGBAImage.get(), image.getNumPixels() * sizeof(uchar4), hipMemcpyDeviceToHost) );


	image.saveRGBAImgOut("blurredImage.jpg");
	}
	}
	*/
}


