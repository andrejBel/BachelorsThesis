#include "MemoryPoolPitched.h"

#include ""

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
namespace processing 
{
	

	MemoryPoolPitched & MemoryPoolPitched::getMemoryPoolPitchedForOutput()
	{
		static MemoryPoolPitched pool(PITCHED_MEMORY_BUFFER_SIZE_OUTPUT);
		return pool;
	}

	MemoryPoolPitched & MemoryPoolPitched::getMemoryPoolPitchedForInput()
	{
		static MemoryPoolPitched pool(PITCHED_MEMORY_BUFFER_SIZE_INPUT);
		return pool;
	}

	MemoryPoolPitched::MemoryPoolPitched(uint memorySize) : memory_(memorySize)
	{
		float * memory = nullptr;
		for (uint i = 0; i < memorySize; i++)
		{
			checkCudaErrors(hipMallocPitch<float>(&memory, &pitch_, (MAX_IMAGE_WIDTH + 300) * sizeof(float), MAX_IMAGE_HEIGHT + 300));
			memory_[i] = memory;
		}
	}

	MemoryPoolPitched::~MemoryPoolPitched()
	{
		for (float * memory : memory_ )
		{
			hipFree(memory);
		}
	}




}
