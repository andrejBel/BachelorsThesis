#include "hip/hip_runtime.h"
#include "KernelSharedMemoryManaged.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "opencv2/core/utility.hpp"

#include <vector>
#include <cstdio>
#include <cmath>
#include <iostream>

#include <thread>
#include <algorithm>

#define IMAD(a, b, c) ( __mul24((a), (b)) + (c) )
#define CONVOLUTIONGPUSHAREDMANAGED(FILTER_W, BLOCK_S, TILE_S)\
case FILTER_W:\
{\
	float * ptr =  (deviceFilters.get() + offset);\
	const int BLOCK_SIZE = BLOCK_S;\
	const int FILTER_WIDTH = FILTER_W;\
	const int TILE_SIZE = TILE_S;\
	static_assert(BLOCK_SIZE - TILE_SIZE >= (FILTER_WIDTH - 1), "Wrong block and tile size, BLOCKSIZE - TILESIZE >= (FILTERWIDTH - 1)");\
	const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);\
	const dim3 gridSize((image.getNumCols() + TILE_SIZE - 1) / TILE_SIZE, (image.getNumRows() + TILE_SIZE - 1) / TILE_SIZE, 1);\
	convolutionGPUSharedManaged<FILTER_WIDTH, BLOCK_SIZE, TILE_SIZE> << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGrayImageIn.get(), resultUnified.get());\
	break;\
}

using namespace std;

namespace processing
{
	/*
	template<typename T, typename int FILTER_WIDTH, typename int BLOCK_SIZE, typename int TILE_SIZE>
	__global__ void convolutionGPUSharedManaged(processing::Filter<T, FILTER_WIDTH> * filter, const int numRows, const int numCols, uchar * inputImage, T * outputImage)
	{
	int2 absoluteImagePosition;
	absoluteImagePosition.x = IMAD(blockIdx.x, TILE_SIZE, threadIdx.x);
	absoluteImagePosition.y = IMAD(blockIdx.y, TILE_SIZE, threadIdx.y);
	int2 sharedPosition;
	sharedPosition.x = absoluteImagePosition.x - (FILTER_WIDTH / 2);
	sharedPosition.y = absoluteImagePosition.y - (FILTER_WIDTH / 2);
	__shared__ uchar shared[BLOCK_SIZE][BLOCK_SIZE];
	int threadX = threadIdx.x;
	int threadY = threadIdx.y;
	sharedPosition.x = min(max(sharedPosition.x, 0), numCols - 1);
	sharedPosition.y = min(max(sharedPosition.y, 0), numRows - 1);
	shared[threadY][threadX] = inputImage[IMAD(sharedPosition.y, numCols, sharedPosition.x)];
	__syncthreads();
	const T* filterV = filter->getFilter();
	T result(0.0);

	if (threadX < TILE_SIZE && threadY < TILE_SIZE && absoluteImagePosition.x < numCols && absoluteImagePosition.y <  numRows)
	{
	#pragma unroll FILTER_WIDTH
	for (int yOffset = 0; yOffset < FILTER_WIDTH; yOffset++)
	{
	#pragma unroll FILTER_WIDTH
	for (int xOffset = 0; xOffset < FILTER_WIDTH; xOffset++)
	{
	result += filterV[IMAD(yOffset, FILTER_WIDTH, xOffset)] * shared[yOffset + threadY][xOffset + threadX];
	}
	}
	outputImage[IMAD(absoluteImagePosition.y, numCols, absoluteImagePosition.x)] = result;
	}
	}
	*/

	template<typename int FILTER_WIDTH, typename int BLOCK_SIZE, typename int TILE_SIZE>
	__global__ void convolutionGPUSharedManaged(float * filter, const int numRows, const int numCols, float * inputImage, float * outputImage)
	{

		int2 absoluteImagePosition;
		absoluteImagePosition.x = IMAD(blockIdx.x, TILE_SIZE, threadIdx.x);
		absoluteImagePosition.y = IMAD(blockIdx.y, TILE_SIZE, threadIdx.y);
		int2 sharedPosition;
		sharedPosition.x = absoluteImagePosition.x - (FILTER_WIDTH / 2);
		sharedPosition.y = absoluteImagePosition.y - (FILTER_WIDTH / 2);
		__shared__ float filterShared[FILTER_WIDTH][FILTER_WIDTH];
		__shared__ float shared[BLOCK_SIZE][BLOCK_SIZE];
		int threadX = threadIdx.x;
		int threadY = threadIdx.y;
		sharedPosition.x = min(max(sharedPosition.x, 0), numCols - 1);
		sharedPosition.y = min(max(sharedPosition.y, 0), numRows - 1);
		shared[threadY][threadX] = inputImage[IMAD(sharedPosition.y, numCols, sharedPosition.x)];
		if (threadX < FILTER_WIDTH && threadY < FILTER_WIDTH)
		{
			filterShared[threadY][threadX] = filter[IMAD(threadY, FILTER_WIDTH, threadX)];
		}
		__syncthreads();
		if (threadX < TILE_SIZE && threadY < TILE_SIZE && absoluteImagePosition.x < numCols && absoluteImagePosition.y <  numRows)
		{

			float result(0.0);
#pragma unroll FILTER_WIDTH
			for (int yOffset = 0; yOffset < FILTER_WIDTH; yOffset++)
			{
#pragma unroll FILTER_WIDTH
				for (int xOffset = 0; xOffset < FILTER_WIDTH; xOffset++)
				{
					result += filterShared[yOffset][xOffset] * shared[yOffset + threadY][xOffset + threadX];
				}
			}
			outputImage[IMAD(absoluteImagePosition.y, numCols, absoluteImagePosition.x)] = result;
		}
	}


	KernelSharedMemoryManaged::KernelSharedMemoryManaged()
	{}


	void KernelSharedMemoryManaged::run(ImageFactory& image, vector<shared_ptr<Filter>>& filters, vector<shared_ptr<float>>& results)
	{
		shared_ptr<float> deviceFilters = makeDeviceFilters(filters);

		// filter allocation and initialization
		float* hostGrayImage = image.getInputGrayPointerFloat();

		shared_ptr<float> deviceGrayImageIn = allocateMemmoryDevice<float>(image.getNumPixels());

		checkCudaErrors(hipMemcpy(deviceGrayImageIn.get(), hostGrayImage, image.getNumPixels() * sizeof(float), hipMemcpyHostToDevice));
		// memory allocation
		shared_ptr<float> deviceGrayImageOut = allocateMemmoryDevice<float>(image.getNumPixels());
		uint offset = 0;
		for (auto& filter : filters)
		{
			shared_ptr<float> resultUnified = allocateManagedMemory<float>(image.getNumPixels());
			checkCudaErrors(hipDeviceSynchronize());
			switch (filter->getWidth())
			{
				CONVOLUTIONGPUSHAREDMANAGED(1, 32, 32)
					CONVOLUTIONGPUSHAREDMANAGED(3, 32, 30)
					CONVOLUTIONGPUSHAREDMANAGED(5, 32, 28)
					CONVOLUTIONGPUSHAREDMANAGED(7, 32, 26)
					CONVOLUTIONGPUSHAREDMANAGED(9, 32, 24)
					CONVOLUTIONGPUSHAREDMANAGED(11, 32, 22)
					CONVOLUTIONGPUSHAREDMANAGED(13, 32, 20)
					CONVOLUTIONGPUSHAREDMANAGED(15, 32, 18)
			default:
				std::cerr << "Filter with width: " << filter->getWidth() << " not supported!" << endl;
				break;
			}
			offset += filter->getSize();
			//checkCudaErrors(hipMemcpy(resultCPU.get(), deviceGrayImageOut.get(), image.getNumPixels() * sizeof(T), hipMemcpyDeviceToHost));
			results.push_back(resultUnified);
		}
		checkCudaErrors(hipDeviceSynchronize());
	}

}

