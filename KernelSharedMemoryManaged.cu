#include "hip/hip_runtime.h"
#include "KernelSharedMemoryManaged.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "processing.h"
#include "Filter.h"

#include "opencv2/core/utility.hpp"


#include <vector>
#include <cstdio>
#include <cmath>
#include <iostream>

#include <thread>
#include <algorithm>

#define IMAD(a, b, c) ( __mul24((a), (b)) + (c) )

using namespace std;

namespace processing
{

	template<typename T, typename int FILTER_WIDTH, typename int BLOCK_SIZE, typename int TILE_SIZE>
	__global__ void convolutionGPUSharedManaged(processing::Filter<T, FILTER_WIDTH> * filter, const int numRows, const int numCols, uchar * inputImage, T * outputImage)
	{
		int2 absoluteImagePosition;
		absoluteImagePosition.x = IMAD(blockIdx.x , TILE_SIZE ,threadIdx.x);
		absoluteImagePosition.y = IMAD(blockIdx.y , TILE_SIZE, threadIdx.y);
		int2 sharedPosition;
		sharedPosition.x = absoluteImagePosition.x - (FILTER_WIDTH / 2);
		sharedPosition.y = absoluteImagePosition.y - (FILTER_WIDTH / 2);
		__shared__ uchar shared[BLOCK_SIZE][BLOCK_SIZE];
		int threadX = threadIdx.x;
		int threadY = threadIdx.y;
		sharedPosition.x = min(max(sharedPosition.x, 0), numCols - 1);
		sharedPosition.y = min(max(sharedPosition.y, 0), numRows - 1);
		shared[threadY][threadX] = inputImage[IMAD(sharedPosition.y , numCols , sharedPosition.x)];
		__syncthreads();
		const T* filterV = filter->getFilter();
		T result(0.0);


		if (threadX < TILE_SIZE && threadY < TILE_SIZE && absoluteImagePosition.x < numCols && absoluteImagePosition.y <  numRows)
		{
#pragma unroll FILTER_WIDTH
			for (int yOffset = 0; yOffset < FILTER_WIDTH; yOffset++)
			{
#pragma unroll FILTER_WIDTH
				for (int xOffset = 0; xOffset < FILTER_WIDTH; xOffset++)
				{
					result += filterV[IMAD(yOffset,FILTER_WIDTH , xOffset)] * shared[yOffset + threadY][xOffset + threadX];
				}
			}
			outputImage[IMAD(absoluteImagePosition.y, numCols ,absoluteImagePosition.x)] = result;
		}
	}

	template<typename T>
	KernelSharedMemoryManaged<T>::KernelSharedMemoryManaged()
	{}

	template<typename T>
	void KernelSharedMemoryManaged<T>::run(ImageFactory& image, vector<shared_ptr<AbstractFilter<T>>>& filters, vector<shared_ptr<T>>& results)
	{
		uint filterCount(filters.size());
		size_t memmoryToAllocateForFiltersOnDevice(0);
		for_each(filters.begin(), filters.end(), [&memmoryToAllocateForFiltersOnDevice](auto& filter) { memmoryToAllocateForFiltersOnDevice += filter->getSize(); });
		shared_ptr<uchar> deviceFilters = allocateMemmoryDevice<uchar>(memmoryToAllocateForFiltersOnDevice);
		uint offset(0);
		int maxFilterWidth = 0;
		for_each(filters.begin(), filters.end(), [&deviceFilters, &offset, &maxFilterWidth](auto& filter)
		{
			filter->copyWholeFilterToDeviceMemory(deviceFilters.get() + offset);
			offset += filter->getSize();
			if (maxFilterWidth < filter->getSize())
			{
				maxFilterWidth = filter->getSize();
			}
		});
		// filter allocation and initialization
		uchar* hostGrayImage = image.getInputGrayPointer();

		shared_ptr<uchar> deviceGrayImageIn = allocateMemmoryDevice<uchar>(image.getNumPixels());
		//std::copy(hostGrayImage, hostGrayImage + image.getNumRows(), deviceGrayImageIn.get());
		checkCudaErrors(hipMemcpy(deviceGrayImageIn.get(), hostGrayImage, image.getNumPixels() * sizeof(uchar), hipMemcpyHostToDevice));
		// memory allocation

		for (auto& filter : filters)
		{
			shared_ptr<T> resultUnified = allocateManagedMemory<T>(image.getNumPixels());
			//hipMemPrefetchAsync(resultUnified.get(), image.getNumPixels() * sizeof(T), device, NULL);
			switch (filter->getWidth())
			{
			case 3:
			{
				Filter<T, 3> * ptr = (Filter<T, 3> *) (deviceFilters.get() + offset);
				const int BLOCK_SIZE = 32;
				const int FILTER_WIDTH = 3;
				const int TILE_SIZE = 30;
				const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
				const dim3 gridSize((image.getNumCols() + TILE_SIZE - 1) / TILE_SIZE, (image.getNumRows() + TILE_SIZE - 1) / TILE_SIZE, 1);
				convolutionGPUSharedManaged<T, FILTER_WIDTH, BLOCK_SIZE, TILE_SIZE> << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGrayImageIn.get(), resultUnified.get());
				break;
			}
			case 5:
			{
				Filter<T, 5> * ptr = (Filter<T, 5> *) (deviceFilters.get() + offset);
				const int BLOCK_SIZE = 32;
				const int FILTER_WIDTH = 5;
				const int TILE_SIZE = 28;
				const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
				const dim3 gridSize((image.getNumCols() + TILE_SIZE - 1) / TILE_SIZE, (image.getNumRows() + TILE_SIZE - 1) / TILE_SIZE, 1);
				static_assert(BLOCK_SIZE - TILE_SIZE >= (FILTER_WIDTH - 1), "Wrong block and tile size, BLOCKSIZE - TILESIZE >= (FILTERWIDTH - 1)");
				convolutionGPUSharedManaged<T, FILTER_WIDTH, BLOCK_SIZE, TILE_SIZE> << <gridSize, blockSize >> > (ptr, image.getNumRows(), image.getNumCols(), deviceGrayImageIn.get(), resultUnified.get());
				break;
			}
			case 7:
			{
				Filter<T, 7> * ptr = (Filter<T, 7> *) (deviceFilters.get() + offset);
				const int BLOCK_SIZE = 32;
				const int FILTER_WIDTH = 7;
				const int TILE_SIZE = 26;
				const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
				const dim3 gridSize((image.getNumCols() + TILE_SIZE - 1) / TILE_SIZE, (image.getNumRows() + TILE_SIZE - 1) / TILE_SIZE, 1);
				convolutionGPUSharedManaged<T, FILTER_WIDTH, BLOCK_SIZE, TILE_SIZE> << <gridSize, blockSize >> >(ptr, image.getNumRows(), image.getNumCols(), deviceGrayImageIn.get(), resultUnified.get());
				break;
			}
			default:
				break;
			}
			offset += filter->getSize();
			//checkCudaErrors(hipMemcpy(resultCPU.get(), deviceGrayImageOut.get(), image.getNumPixels() * sizeof(T), hipMemcpyDeviceToHost));
			
			results.push_back(resultUnified);
		}
		checkCudaErrors(hipDeviceSynchronize());
		
	}

}

