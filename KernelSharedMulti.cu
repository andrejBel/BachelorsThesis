#include "hip/hip_runtime.h"
#include "KernelSharedMulti.h"


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "opencv2/core/utility.hpp"

#include "MemoryPoolPinned.h"
#include <vector>
#include <cstdio>
#include <cmath>
#include <iostream>

#include <algorithm>
#include <type_traits>
#include <utility>

#define MUL(a, b) __mul24(a, b)
#define IMAD(a, b, c) ( __mul24((a), (b)) + (c) )
#define CEIL(a, b) ((a + b - 1) / b)
#define ROUNDUP(a, b) (((a + b - 1) / b) * b)

using namespace std;
namespace processing
{
#define  CONVOLUTIONSHAREDMULTI(FILTERWIDTH, BLOCKSIZEX, BLOCKSIZEY, TILESIZEX, TILESIZEY) \
					case FILTERWIDTH: \
					{ \
						const short MAX_SMALL_TILE_DIMENION_X = 2;\
						const short MAX_SMALL_TILE_DIMENION_Y = 2;\
						int colsForGridX = CEIL(numCols, MAX_SMALL_TILE_DIMENION_X); \
						int rowsForGridY = CEIL(numRows, MAX_SMALL_TILE_DIMENION_Y); \
						const int FILTER_WIDTH = FILTERWIDTH; \
						const int BLOCK_SIZE_X = BLOCKSIZEX; \
						const int BLOCK_SIZE_Y = BLOCKSIZEY; \
						const int TILE_SIZE_X = TILESIZEX; \
						const int TILE_SIZE_Y = TILESIZEY; \
						const dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y); \
						const dim3 gridSize((colsForGridX + TILE_SIZE_X - 1) / TILE_SIZE_X, (rowsForGridY + TILE_SIZE_Y - 1) / TILE_SIZE_Y, 1); \
						if (makezeros)\
						{\
							convolutionGPUSharedIncompleteBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, true> << <gridSize, blockSize >> > (inputImagesDevice[k], deviceGrayImageOut, pitchInput / sizeof(float), pitchOutput / sizeof(float), k);\
						}\
						else\
						{\
							convolutionGPUSharedIncompleteBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, false> << <gridSize, blockSize >> > (inputImagesDevice[k], deviceGrayImageOut, pitchInput / sizeof(float), pitchOutput / sizeof(float), k);\
						} \
						break; \
					}

#define CONVOLUTIONSHAREDMULTIFULL(FILTERWIDTH, BLOCKSIZEX, BLOCKSIZEY) \
					case FILTERWIDTH: \
					{ \
						const short MAX_SMALL_TILE_DIMENION_X = 3;\
						const short MAX_SMALL_TILE_DIMENION_Y = 3;\
						int colsForGridX = CEIL(numCols, MAX_SMALL_TILE_DIMENION_X); \
						int rowsForGridY = CEIL(numRows, MAX_SMALL_TILE_DIMENION_Y); \
						const int FILTER_WIDTH = FILTERWIDTH; \
						const int BLOCK_SIZE_X = BLOCKSIZEX; \
						const int BLOCK_SIZE_Y = BLOCKSIZEY; \
						const int TILE_SIZE_X = 3; \
						const int TILE_SIZE_Y = 3; \
						const dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y); \
						const dim3 gridSize((colsForGridX + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (rowsForGridY + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, 1); \
						if (makezeros) \
						{\
							convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, true> << <gridSize, blockSize >> > (inputImagesDevice[k], deviceGrayImageOut, pitchInput / sizeof(float), pitchOutput / sizeof(float), k); \
						}\
						else \
						{\
							convolutionGPUSharedFullBlock< FILTER_WIDTH, BLOCK_SIZE_X, BLOCK_SIZE_Y, TILE_SIZE_X, TILE_SIZE_Y, false> << <gridSize, blockSize >> > (inputImagesDevice[k], deviceGrayImageOut, pitchInput / sizeof(float), pitchOutput / sizeof(float), k); \
						}\
						break; \
					}


	template<typename int FILTER_WIDTH, typename int BLOCK_SIZE_X, typename int BLOCK_SIZE_Y, typename int TILE_SIZE_X, typename int TILE_SIZE_Y, typename bool MAKEZEROS, typename int MAX_SMALL_TILE_DIMENION_X = 2, typename int MAX_SMALL_TILE_DIMENION_Y = 2>
	__global__ void convolutionGPUSharedIncompleteBlock(float * __restrict__ inputImage, float * __restrict__  outputImage, int inputPitch, int outputPitch, const short filterIndex)
	{
		__shared__ float shared[BLOCK_SIZE_Y * MAX_SMALL_TILE_DIMENION_Y][BLOCK_SIZE_X * MAX_SMALL_TILE_DIMENION_X];
		int threadX = MUL(threadIdx.x, MAX_SMALL_TILE_DIMENION_X);
		int threadY = MUL(threadIdx.y, MAX_SMALL_TILE_DIMENION_Y);
		int2 absoluteImagePosition;
		absoluteImagePosition.x = IMAD(blockIdx.x, TILE_SIZE_X, threadIdx.x) * MAX_SMALL_TILE_DIMENION_X;
		absoluteImagePosition.y = IMAD(blockIdx.y, TILE_SIZE_Y, threadIdx.y) * MAX_SMALL_TILE_DIMENION_Y;

#pragma unroll MAX_SMALL_TILE_DIMENION_Y
		for (int i = 0; i < MAX_SMALL_TILE_DIMENION_Y; i++)
		{
			* ((float2 *)&shared[threadY + i][threadX]) = *(float2 *)(inputImage + IMAD(absoluteImagePosition.y + i, inputPitch, absoluteImagePosition.x));
		}
		__syncthreads();
		if (threadX < TILE_SIZE_X * MAX_SMALL_TILE_DIMENION_X  && threadY < TILE_SIZE_Y * MAX_SMALL_TILE_DIMENION_Y)
		{
			float results[MAX_SMALL_TILE_DIMENION_X * MAX_SMALL_TILE_DIMENION_Y] = { 0.0 };
			float filterValue;
#pragma unroll FILTER_WIDTH
			for (int yOffset = 0; yOffset < FILTER_WIDTH; yOffset++)
			{
#pragma unroll FILTER_WIDTH
				for (int xOffset = 0; xOffset < FILTER_WIDTH; xOffset++)
				{
					filterValue = FILTERCUDA[yOffset*FILTER_WIDTH + xOffset + FILTER_WIDTH * FILTER_WIDTH * filterIndex];
#pragma unroll MAX_SMALL_TILE_DIMENION_Y
					for (int i = 0; i < MAX_SMALL_TILE_DIMENION_Y; i++)
					{
#pragma unroll MAX_SMALL_TILE_DIMENION_X
						for (int j = 0; j < MAX_SMALL_TILE_DIMENION_X; j++)
						{
							results[IMAD(i, MAX_SMALL_TILE_DIMENION_Y, j)] += filterValue * shared[yOffset + threadY + i][xOffset + threadX + j];
						}

					}
				}
			}
			/*
#pragma unroll MAX_SMALL_TILE_DIMENION_Y
			for (int i = 0; i < MAX_SMALL_TILE_DIMENION_Y; i++)
			{
				//* ((float2 *)(outputImage + IMAD(absoluteImagePosition.y + i, outputPitch, absoluteImagePosition.x))) = *((float2 *)(&results[MUL(i, MAX_SMALL_TILE_DIMENION_Y)]));

#pragma unroll MAX_SMALL_TILE_DIMENION_X
				for (int j = 0; j < MAX_SMALL_TILE_DIMENION_X; j++)
				{
					outputImage[IMAD(absoluteImagePosition.y + i, outputPitch, absoluteImagePosition.x + j)] += results[i *MAX_SMALL_TILE_DIMENION_Y + j];
				}
			}
			*/
			if (MAKEZEROS)
			{
				*((float2 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x))) = *((float2*)&results[0]);
				*((float2 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x))) = *((float2*)&results[2]);
			}
			else
			{
				float2 fromGlobal = *((float2 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x)));
				fromGlobal.x += results[0];
				fromGlobal.y += results[1];
				*((float2 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x))) = fromGlobal;
				fromGlobal = *((float2 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x)));
				fromGlobal.x += results[2];
				fromGlobal.y += results[3];
				*((float2 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x))) = fromGlobal;
			}
		}
	}

	template<typename int FILTER_WIDTH, typename int BLOCK_SIZE_X, typename int BLOCK_SIZE_Y, typename int TILE_SIZE_X, typename int TILE_SIZE_Y, typename bool MAKEZEROS>
	__global__ void convolutionGPUSharedFullBlock(float * __restrict__ inputImage, float *__restrict__ outputImage, const int inputPitch, const int outputPitch, const short filterIndex)
	{
		float results[TILE_SIZE_X * TILE_SIZE_Y];
		int2 absoluteImagePosition;
		absoluteImagePosition.x = (blockIdx.x *  blockDim.x + threadIdx.x) * TILE_SIZE_X;
		absoluteImagePosition.y = (blockIdx.y * blockDim.y + threadIdx.y) * TILE_SIZE_Y;
		int2 positionShared;
		positionShared.x = blockIdx.x * BLOCK_SIZE_X * TILE_SIZE_X;
		positionShared.y = blockIdx.y * BLOCK_SIZE_Y * TILE_SIZE_Y;
		__shared__ float shared[BLOCK_SIZE_Y*TILE_SIZE_Y + (FILTER_WIDTH - 1) + TILE_SIZE_Y][BLOCK_SIZE_X *TILE_SIZE_X + (FILTER_WIDTH - 1) + TILE_SIZE_X];
		int threadX = threadIdx.x * TILE_SIZE_X;
		int threadY = threadIdx.y * TILE_SIZE_Y;
		for (int j = threadY; j < FILTER_WIDTH - 1 + BLOCK_SIZE_Y * TILE_SIZE_Y; j += BLOCK_SIZE_Y * TILE_SIZE_Y)
		{
			for (int i = threadX; i < FILTER_WIDTH - 1 + BLOCK_SIZE_X * TILE_SIZE_X; i += BLOCK_SIZE_X * TILE_SIZE_X)
			{
#pragma unroll TILE_SIZE_Y
				for (int k = 0; k < TILE_SIZE_Y; k++)
				{

					*((float3 *)&shared[j + k][i]) = *(float3 *)(inputImage + IMAD(positionShared.y + j + k, inputPitch, positionShared.x + i));
				}
			}
		}
		__syncthreads();
#pragma unroll TILE_SIZE_Y
		for (int k = 0; k < TILE_SIZE_Y; k++)
		{
#pragma unroll TILE_SIZE_X
			for (int l = 0; l < TILE_SIZE_X; l++)
			{
				results[k * TILE_SIZE_Y + l] = 0.0;
			}
		}
		
		float filterValue;
#pragma unroll FILTER_WIDTH
		for (int yOffset = 0; yOffset < FILTER_WIDTH; yOffset++)
		{
#pragma unroll FILTER_WIDTH
			for (int xOffset = 0; xOffset < FILTER_WIDTH; xOffset++)
			{
				filterValue = FILTERCUDA[yOffset*FILTER_WIDTH + xOffset + FILTER_WIDTH * FILTER_WIDTH * filterIndex];
#pragma unroll TILE_SIZE_Y
				for (int k = 0; k < TILE_SIZE_Y; k++)
				{
#pragma unroll TILE_SIZE_X
					for (int l = 0; l < TILE_SIZE_X; l++)
					{
						results[k * TILE_SIZE_Y + l] += filterValue * shared[yOffset + threadY + k][xOffset + threadX + l];
					}
				}
			}
		}
		if (MAKEZEROS)
		{
			*((float3 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x))) = *((float3*)&results[0]);
			*((float3 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x))) = *((float3*)&results[3]);
			*((float3 *)(outputImage + IMAD(absoluteImagePosition.y + 2, outputPitch, absoluteImagePosition.x))) = *((float3*)&results[6]);
		}
		else
		{
			float3 fromGlobal = *((float3 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x)));
			fromGlobal.x += results[0];
			fromGlobal.y += results[1];
			fromGlobal.z += results[2];
			*((float3 *)(outputImage + IMAD(absoluteImagePosition.y, outputPitch, absoluteImagePosition.x))) = fromGlobal;
			fromGlobal = *((float3 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x)));
			fromGlobal.x += results[3];
			fromGlobal.y += results[4];
			fromGlobal.z += results[5];
			*((float3 *)(outputImage + IMAD(absoluteImagePosition.y + 1, outputPitch, absoluteImagePosition.x))) = fromGlobal;
			fromGlobal = *((float3 *)(outputImage + IMAD(absoluteImagePosition.y + 2, outputPitch, absoluteImagePosition.x)));
			fromGlobal.x += results[6];
			fromGlobal.y += results[7];
			fromGlobal.z += results[8];
			*((float3 *)(outputImage + IMAD(absoluteImagePosition.y + 2, outputPitch, absoluteImagePosition.x))) = fromGlobal;
		}

	}

	KernelSharedMulti::KernelSharedMulti() : MultiRunnable()
	{
	}

	void KernelSharedMulti::run(vector<shared_ptr<ImageFactory>>& images, vector<vector<shared_ptr<Filter>>>& filters, vector<shared_ptr<float>>& results)
	{
		
		int maxImageWidth = 0;
		int maxImageHeight = 0;
		for_each(images.begin(), images.end(), [&maxImageWidth, &maxImageHeight](shared_ptr<ImageFactory> image)
		{
			if (image->getNumCols() > maxImageWidth)
			{
				maxImageWidth = image->getNumCols();
			}
			if (image->getNumRows() > maxImageHeight)
			{
				maxImageHeight = image->getNumRows();
			}
		});
		MemoryPoolPitched::getMemoryPoolPitchedForInput().realoc(maxImageWidth, maxImageHeight);
		MemoryPoolPitched::getMemoryPoolPitchedForOutput().realoc(maxImageWidth, maxImageHeight);

		size_t pitchInput = MemoryPoolPitched::getMemoryPoolPitchedForInput().getPitch();
		size_t pitchOutput = MemoryPoolPitched::getMemoryPoolPitchedForOutput().getPitch();
		int numCols = images[0]->getNumCols(); //x
		int numRows = images[0]->getNumRows(); //y

		vector<float *> inputImagesDevice;
		for (int i = 0; i < PITCHED_MEMORY_BUFFER_SIZE_INPUT; i++)
		{
			inputImagesDevice.push_back(MemoryPoolPitched::getMemoryPoolPitchedForInput().getMemory()[i]);
		}
		for (int i = 0; i < PITCHED_MEMORY_BUFFER_SIZE_OUTPUT - 1; i++)
		{
			inputImagesDevice.push_back(MemoryPoolPitched::getMemoryPoolPitchedForOutput().getMemory()[i]);
		}
		float * deviceGrayImageOut = MemoryPoolPitched::getMemoryPoolPitchedForOutput().getMemory()[PITCHED_MEMORY_BUFFER_SIZE_OUTPUT - 1];



		int imageSize = static_cast<int>(images.size());
		size_t filterGroupCount = filters.size();
		vector<vector<shared_ptr<float>>> partialResults(filterGroupCount);



		for (int i = 0; i < imageSize; i += inputImagesDevice.size())
		{
			int startOfImages = i;
			int endOfImages = std::min(i + static_cast<int>(inputImagesDevice.size()) - 1, imageSize - 1);
			int usedImages = endOfImages - startOfImages + 1;

			for (int indexImages = startOfImages, indexDeviceInput = 0; indexImages <= endOfImages; ++indexImages, ++indexDeviceInput)
			{
				checkCudaErrors(hipMemcpy2D(inputImagesDevice[indexDeviceInput], pitchInput, images[indexImages]->getInputGrayPointerFloat(), numCols * sizeof(float), numCols * sizeof(float), numRows, hipMemcpyHostToDevice));
			}

			for (size_t j = 0; j < filterGroupCount; ++j)
			{
				//checkCudaErrors(hipMemset2D(deviceGrayImageOut, pitchOutput, 0, numCols * sizeof(float), numRows));
				vector<shared_ptr<Filter>>& groupFilters = filters[j];
				int filterWidth = groupFilters[0]->getWidth();
				int sizeOfFilter = filterWidth* filterWidth;
				shared_ptr<float> memoryForFilters = shared_ptr<float>(new float[sizeOfFilter * usedImages], [](float * ptr) { delete[] ptr; });
				for (int offset = 0, index = startOfImages; index <= endOfImages; ++index, offset += (filterWidth * filterWidth))
				{
					std::copy(groupFilters[index]->getFilter(), groupFilters[index]->getFilter() + sizeOfFilter, memoryForFilters.get() + offset);
				}
				checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(FILTERCUDA), memoryForFilters.get(), sizeof(float) * sizeOfFilter * usedImages));
				int xlen = numCols - (filterWidth - 1);
				int ylen = numRows - (filterWidth - 1);
				
				for (int k = 0; k < usedImages; k++)
				{
					bool makezeros = false;
					if (k == 0) 
					{
						makezeros = true;
					}
					switch (filterWidth)
					{
						/*
						CONVOLUTIONSHAREDMULTI(1, 32, 16, 32, 16);
						CONVOLUTIONSHAREDMULTI(3, 32, 16, 31, 15);
						CONVOLUTIONSHAREDMULTI(5, 32, 16, 30, 14);
						CONVOLUTIONSHAREDMULTI(7, 32, 32, 29, 29)
						CONVOLUTIONSHAREDMULTI(9, 32, 32, 28, 28)
						CONVOLUTIONSHAREDMULTI(11, 32, 32, 27, 27)
						CONVOLUTIONSHAREDMULTI(13, 32, 32, 26, 26)
						CONVOLUTIONSHAREDMULTI(15, 32, 32, 25, 25)
							*/
							
						CONVOLUTIONSHAREDMULTI(1, 32, 16, 32, 16)
						CONVOLUTIONSHAREDMULTI(3, 32, 16, 31, 15)
						CONVOLUTIONSHAREDMULTI(5, 32, 16, 30, 14)
						CONVOLUTIONSHAREDMULTI(7, 32, 32, 29, 29)
						CONVOLUTIONSHAREDMULTIFULL(9, 32, 8)
						CONVOLUTIONSHAREDMULTIFULL(11, 32, 8)
						CONVOLUTIONSHAREDMULTIFULL(13, 32, 6);
						CONVOLUTIONSHAREDMULTIFULL(15, 32, 8);
						
					default:
						std::cerr << "Filter with width: " << filterWidth << " not supported!" << endl;
						break;
					}
				}
				shared_ptr<float> resultCPU = MemoryPoolPinned::getMemoryPoolPinnedForOutput().acquireMemory(xlen*ylen, true);
				checkCudaErrors(hipMemcpy2D(resultCPU.get(), xlen * sizeof(float), deviceGrayImageOut, pitchOutput, xlen * sizeof(float), ylen, hipMemcpyDeviceToHost));
				checkCudaErrors(hipDeviceSynchronize());
				partialResults[j].push_back(resultCPU);
			}
		}

		for (size_t j = 0; j < filterGroupCount; ++j)
		{
			int filterWidth = filters[j][0]->getWidth();
			int xlen = numCols - (filterWidth - 1);
			int ylen = numRows - (filterWidth - 1);
			size_t range = xlen*ylen;
			shared_ptr<float> result = partialResults[j][0];
			float* resultPointer = result.get();
			for (int i = 1; i < partialResults[j].size(); ++i)
			{
				float * nextResult = partialResults[j][i].get();
				for (size_t k = 0; k < range; ++k)
				{
					resultPointer[k] += nextResult[k];
				}
			}
			results.push_back(result);
		}

	}

}